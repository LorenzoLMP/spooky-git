#include "hip/hip_runtime.h"
#include <array>
#include <complex>
#include <iostream>
#include <random>
#include <stdexcept>
#include <vector>
#include "cufft_routines.hpp"
#include <hip/hip_runtime.h>
#include "common.hpp"
#include "cublas_routines.hpp"
// #include "cuda_kernels.hpp"
#include "cuda_kernels_generic.hpp"
// #include <hip/hip_runtime.h>




int test_forward_inverse_transform(){
    // size_t grid.NX = 256;
    // size_t grid.NY = 128;
    // size_t grid.NZ = 64;
    // dim_t  grid.FFT_SIZE = {grid.NX, grid.NY, grid.NZ};
    // size_t ntotal = grid.FFT_SIZE[0] * grid.FFT_SIZE[1] * grid.FFT_SIZE[2];
    // size_t grid.NTOTAL_COMPLEX = grid.FFT_SIZE[0] * grid.FFT_SIZE[1] * ((grid.FFT_SIZE[2] / 2) + 1);
    int Niter = 50;
    // extern int dimGrid, dimBlock;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int success = 1; // fail



    cpudata_t cpu_r_data((size_t) 2*grid.NTOTAL_COMPLEX);
    cpudata_t cpu_r_data_out((size_t) 2*grid.NTOTAL_COMPLEX);

    unsigned int idx;
    for (int i = 0; i < grid.NX; i++){
        for (int j = 0; j < grid.NY; j++){
            for (int k = 0; k < grid.NZ; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                cpu_r_data[idx] = idx;
            }
        }
    }
    for (int i = 0; i < grid.NX; i++){
        for (int j = 0; j < grid.NY; j++){
            for (int k = grid.NZ; k < grid.NZ+2; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                cpu_r_data[idx] = 0.0;
            }
        }
    }

    std::printf("Input array:\n");
    for (int i = 0; i < 2; i++){
        for (int j = 0; j < 2; j++){
            for (int k = 0; k < 2; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                std::printf("v[%d] %f \n", idx, cpu_r_data[idx]);
            }
        }
    }
    std::printf("=====\n");


    // Create device data arrays
    void *c_data;
    CUDA_RT_CALL(hipMalloc(&c_data, (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX));
    std::printf("array size (in MiB): %f \n",(float) (sizeof(data_type) * grid.NTOTAL_COMPLEX/1e6));
    // Create pointer to complex array to store real data
    hipfftDoubleReal *r_data = (scalar_type *) c_data;
    // Copy input data to GPUs
    CUDA_RT_CALL(hipMemcpy(r_data, cpu_r_data.data(), (size_t) sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX, hipMemcpyHostToDevice));

    // init plans
    init_plan(grid.FFT_SIZE);

    hipEventRecord(start);
    // Do forward and inverse transform
    for (int ii = 0; ii < Niter; ii++) {
        r2c_fft(r_data, c_data);
        c2r_fft(c_data, r_data);
    }
    hipEventRecord(stop);

    // Copy output data to CPU
    CUDA_RT_CALL(hipMemcpy(cpu_r_data_out.data(), r_data, sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX, hipMemcpyDeviceToHost));


    std::printf("Output array:\n");
    for (int i = 0; i < 2; i++){
        for (int j = 0; j < 2; j++){
            for (int k = 0; k < 2; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                std::printf("v[%d] %f \n", idx, cpu_r_data_out[idx]);
            }
        }
    }
    std::printf("=====\n");

    CUDA_RT_CALL(hipFree(c_data));

    finish_cufft();


    // verify results
    double error{};
    double ref{};
    for (int i = 0; i < grid.NX; i++){
        for (int j = 0; j < grid.NY; j++){
            for (int k = 0; k < grid.NZ; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                error += std::norm(cpu_r_data[idx] - cpu_r_data_out[idx]);
                ref += std::norm(cpu_r_data_out[idx]);
            }
        }
    }

    double l2_error = (ref == 0.0) ? std::sqrt(error) : std::sqrt(error) / std::sqrt(ref);
    if (l2_error < 1e-12) {
        std::cout << "PASSED with L2 error = " << l2_error << std::endl;
        success = 0; // success
    } else {
        std::cout << "FAILED with L2 error = " << l2_error << std::endl;
    };

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::printf("Elapsed time (in s): %.5f \t Approx time per FFT (in ms): %.5f \n",milliseconds/1000, 0.5*milliseconds/Niter);
    float gflops = 1e-9*5*(grid.NTOTAL_COMPLEX)*log2(grid.NTOTAL_COMPLEX)/(0.5*1e-3*milliseconds/Niter);
    std::printf("Average GFlop/s (per Fourier transform) %.2f\n",gflops);
    

    return success;
};

/*
void test_do_multiplications() {
    // init_plan(grid.FFT_SIZE);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // hipError_t devsyncherr;

    // Do forward and inverse transform
    int Niter=100;

    cpudata_t cpu_r_data1((size_t) 2*grid.NTOTAL_COMPLEX);
    cpudata_t cpu_r_data2((size_t) 2*grid.NTOTAL_COMPLEX);

    unsigned int idx;
    for (int i = 0; i < grid.NX; i++){
        for (int j = 0; j < grid.NY; j++){
            for (int k = 0; k < grid.NZ; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                cpu_r_data1[idx] = idx;
                cpu_r_data2[idx] = 2.0*idx;
            }
        }
    }
    for (int i = 0; i < grid.NX; i++){
        for (int j = 0; j < grid.NY; j++){
            for (int k = grid.NZ; k < grid.NZ+2; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                cpu_r_data1[idx] = 0.0;
                cpu_r_data2[idx] = 0.0;
            }
        }
    }
    cpu_r_data1[0] = 1.0;
    cpu_r_data2[0] = 1.0;

    std::printf("Input array:\n");
    for (int i = 0; i < 2; i++){
        for (int j = 0; j < 2; j++){
            for (int k = 0; k < 2; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                std::printf("v1[%d]= %f \t v2[%d]= %f \n", idx, cpu_r_data1[idx], idx, cpu_r_data2[idx]);
                // std::printf("v2[%d] %f \n", idx, cpu_r_data2[idx]);
            }
        }
    }
    std::printf("=====\n");

    // Create device data arrays
    scalar_type *dev_data1;
    scalar_type *dev_data2;
    CUDA_RT_CALL(hipMalloc(&dev_data1, (size_t) sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX));
    CUDA_RT_CALL(hipMalloc(&dev_data2, (size_t) sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX));
    std::printf("array size (in MiB): %f \n",(float) (sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX/1e6));

    // Copy input data to GPUs
    CUDA_RT_CALL(hipMemcpy(dev_data1, cpu_r_data1.data(), (size_t) sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX, hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(dev_data2, cpu_r_data2.data(), (size_t) sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX, hipMemcpyHostToDevice));

    float milliseconds = 0;
    int numElements = 2*grid.NTOTAL_COMPLEX;
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

/////////////////////////////////////////////////////
    // using THRUST
/////////////////////////////////////////////////////

    hipEventRecord(start);
    for (int ii = 0; ii < Niter; ii++) {
        // std::printf("iter %d \n", ii);
        // thrust::transform(thrust::device_pointer_cast(d_farray_r[0]), thrust::device_pointer_cast(d_farray_r[0])+2*grid.NTOTAL_COMPLEX, thrust::device_pointer_cast(d_farray_r[1]), thrust::device_pointer_cast(d_farray_r[0]), thrust::multiplies<scalar_type>());
        // for (int n = 0 ; n < num_fields ; n++) {
        //     r2c_fft(d_farray_r[n], d_farray[n]);
        //     // c2r_fft(d_farray[n], d_farray_r[n]);
        // }

        // this operation does pointwise v1*v2 operation and stores the result in v1
        thrust::transform(thrust::device_pointer_cast(dev_data1), thrust::device_pointer_cast(dev_data1)+2*grid.NTOTAL_COMPLEX, thrust::device_pointer_cast(dev_data2), thrust::device_pointer_cast(dev_data1), thrust::multiplies<scalar_type>());
        // divide back
        thrust::transform(thrust::device_pointer_cast(dev_data1), thrust::device_pointer_cast(dev_data1)+2*grid.NTOTAL_COMPLEX, thrust::device_pointer_cast(dev_data2), thrust::device_pointer_cast(dev_data1), thrust::divides<scalar_type>());


    }
    hipEventRecord(stop);

    hipDeviceSynchronize();
    hipEventElapsedTime(&milliseconds, start, stop);
    std::printf("Thrust elapsed time (in s): %.5f \t Approx time per multiply (in ms): %.5f \n",milliseconds/1000, milliseconds/Niter);


    ///////////////////////////////////////////////////
    // same operation as before but with CUBLAS
    ///////////////////////////////////////////////////

    init_cublas();
    hipblasStatus_t stat;
    scalar_type *scratch;
    CUDA_RT_CALL(hipMalloc(&scratch, (size_t) sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX));
    // hipblasSideMode_t mode = HIPBLAS_SIDE_RIGHT;
    // int n = 2*grid.NTOTAL_COMPLEX;
    // int m = 1;
    // int lda = 2*grid.NTOTAL_COMPLEX;
    // int incx = 1;
    // int ldc = 2*grid.NTOTAL_COMPLEX;

    hipblasSideMode_t mode = HIPBLAS_SIDE_LEFT;
    int m = numElements;
    int n = 1;
    int lda = numElements;
    int incx = 1;
    int ldc = numElements;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
            threadsPerBlock);
    hipEventRecord(start);
    for (int ii = 0; ii < Niter; ii++) {
        // std::printf("iter %d \n", ii);
        // this operation does pointwise v1*v2 operation and stores the result in v1
        stat = hipblasDdgmm(handle0, mode, m, n, (double *)dev_data1, lda, (double *)dev_data2, incx, (double *)dev_data1, ldc);
        // std::cout << (int)stat << std::endl;
        // if (stat != HIPBLAS_STATUS_SUCCESS) std::printf("ERROR \n");
        // compute 1/v2
        RvectorReciprocal<<<blocksPerGrid, threadsPerBlock>>>((double *)dev_data2, (double *)scratch, numElements);
        //divide back
        stat = hipblasDdgmm(handle0, mode, m, n, (double *)dev_data1, lda, (double *)scratch, incx, (double *)dev_data1, ldc);
    }
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&milliseconds, start, stop);
    std::printf("CUBLAS elapsed time (in s): %.5f \t Approx time per multiply (in ms): %.5f \n",milliseconds/1000, milliseconds/Niter);
    finish_cublas();
    CUDA_RT_CALL(hipFree(scratch));


    //////////////////////////////////////////////////////////////
    // same operation as before but with custom kernels
    //////////////////////////////////////////////////////////////

    hipEventRecord(start);
    for (int ii = 0; ii < Niter; ii++) {
        // this operation does pointwise v1*v2 operation and stores the result in v1
        RRvectorMultiply<<<blocksPerGrid, threadsPerBlock>>>((double *)dev_data1, (double *)dev_data2, (double *)dev_data1, (double) 1.0, numElements);
        RRvectorDivide<<<blocksPerGrid, threadsPerBlock>>>((double *)dev_data1, (double *)dev_data2, (double *)dev_data1, (double) 1.0, numElements);
    }
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&milliseconds, start, stop);
    std::printf("custom kernels elapsed time (in s): %.5f \t Approx time per multiply (in ms): %.5f \n",milliseconds/1000, milliseconds/Niter);



    // Copy output data to CPU
    CUDA_RT_CALL(hipMemcpy(cpu_r_data1.data(), dev_data1, sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX, hipMemcpyDeviceToHost));
    CUDA_RT_CALL(hipMemcpy(cpu_r_data2.data(), dev_data2, sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX, hipMemcpyDeviceToHost));


    std::printf("Output array:\n");
    for (int i = 0; i < 2; i++){
        for (int j = 0; j < 2; j++){
            for (int k = 0; k < 2; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                std::printf("v1[%d]= %f \t v2[%d]= %f \n", idx, cpu_r_data1[idx], idx, cpu_r_data2[idx]);
            }
        }
    }
    std::printf("=====\n");

    CUDA_RT_CALL(hipFree(dev_data1));
    CUDA_RT_CALL(hipFree(dev_data2));

    // finish_cufft();


    // std::printf("kmax: %.5f \n",wavevector.kmax);
    // return EXIT_SUCCESS;

};



void test_axpy() {
    // init_plan(grid.FFT_SIZE);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // hipError_t devsyncherr;

    // Do forward and inverse transform
    int Niter=1;

    cpudata_t cpu_r_data1((size_t) 2*grid.NTOTAL_COMPLEX);
    cpudata_t cpu_r_data2((size_t) 2*grid.NTOTAL_COMPLEX);


    unsigned int idx;
    for (int i = 0; i < grid.NX; i++){
        for (int j = 0; j < grid.NY; j++){
            for (int k = 0; k < grid.NZ; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                cpu_r_data1[idx] = idx;
                cpu_r_data2[idx] = 2.0*idx;
            }
        }
    }
    for (int i = 0; i < grid.NX; i++){
        for (int j = 0; j < grid.NY; j++){
            for (int k = grid.NZ; k < grid.NZ+2; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                cpu_r_data1[idx] = 0.0;
                cpu_r_data2[idx] = 0.0;
            }
        }
    }
    cpu_r_data1[0] = 1.0;
    cpu_r_data2[0] = 1.0;

    std::printf("Input array:\n");
    for (int i = 0; i < 2; i++){
        for (int j = 0; j < 2; j++){
            for (int k = 0; k < 2; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                std::printf("v1[%d]= %f \t v2[%d]= %f \n", idx, cpu_r_data1[idx], idx, cpu_r_data2[idx]);
                // std::printf("v2[%d] %f \n", idx, cpu_r_data2[idx]);
            }
        }
    }
    std::printf("=====\n");

    // Create device data arrays
    scalar_type *dev_data1;
    scalar_type *dev_data2;
    CUDA_RT_CALL(hipMalloc(&dev_data1, (size_t) sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX));
    CUDA_RT_CALL(hipMalloc(&dev_data2, (size_t) sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX));
    std::printf("array size (in MiB): %f \n",(float) (sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX/1e6));
    scalar_type *scratch;
    CUDA_RT_CALL(hipMalloc(&scratch, (size_t) sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX));

    // Copy input data to GPUs
    CUDA_RT_CALL(hipMemcpy(dev_data1, cpu_r_data1.data(), (size_t) sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX, hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(dev_data2, cpu_r_data2.data(), (size_t) sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX, hipMemcpyHostToDevice));

    float milliseconds = 0;
    int numElements = 2*grid.NTOTAL_COMPLEX;
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    scalar_type scale = 1.0;

    //////////////////////////////////////////////////////////////
    // kernels with double vectors
    //////////////////////////////////////////////////////////////

    hipEventRecord(start);
    for (int ii = 0; ii < Niter; ii++) {
        // using doubles
        axpyDouble<<<blocksPerGrid, threadsPerBlock>>>( (double *)dev_data1, (double *)dev_data2, (double *)scratch, scale, (scalar_type) 1.0, numElements);
    }
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&milliseconds, start, stop);
    std::printf("double vectors elapsed time (in s): %.5f \t Approx time per multiply (in ms): %.5f \n",milliseconds/1000, milliseconds/Niter);

    //////////////////////////////////////////////////////////////
    // kernels with complex vectors
    //////////////////////////////////////////////////////////////

    hipEventRecord(start);
    for (int ii = 0; ii < Niter; ii++) {
        // using complex
        axpyComplex<<<blocksPerGrid, threadsPerBlock>>>( (data_type *)dev_data1, (data_type *)dev_data2, (data_type *)scratch, scale, (scalar_type) 1.0, grid.NTOTAL_COMPLEX);
    }
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&milliseconds, start, stop);
    std::printf("complex vectors elapsed time (in s): %.5f \t Approx time per multiply (in ms): %.5f \n",milliseconds/1000, milliseconds/Niter);

    // Copy output data to CPU
    CUDA_RT_CALL(hipMemcpy(cpu_r_data1.data(), dev_data1, sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX, hipMemcpyDeviceToHost));
    CUDA_RT_CALL(hipMemcpy(cpu_r_data2.data(), dev_data2, sizeof(scalar_type) * 2 * grid.NTOTAL_COMPLEX, hipMemcpyDeviceToHost));


    std::printf("Output array:\n");
    for (int i = 0; i < 2; i++){
        for (int j = 0; j < 2; j++){
            for (int k = 0; k < 2; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                std::printf("v1[%d]= %f \t v2[%d]= %f \n", idx, cpu_r_data1[idx], idx, cpu_r_data2[idx]);
            }
        }
    }
    std::printf("=====\n");

    CUDA_RT_CALL(hipFree(dev_data1));
    CUDA_RT_CALL(hipFree(dev_data2));

    // finish_cufft();


    // std::printf("kmax: %.5f \n",wavevector.kmax);
    // return EXIT_SUCCESS;

};*/
