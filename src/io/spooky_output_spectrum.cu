#include "common.hpp"
#include "cufft_routines.hpp"
#include "cuda_kernels_generic.hpp"
// #include "../libs/HighFive/include/highfive/highfive.hpp"
// #include <highfive/highfive.hpp>
#include "fields.hpp"
// #include "hdf5_io.hpp"
// #include "output_timevar.hpp"
// #include "spooky_outputs.hpp"
#include <iostream>
#include <fstream>
#include <iomanip>
#include "parameters.hpp"
#include "inputoutput.hpp"
#include "timestepping.hpp"
#include "supervisor.hpp"

void writeSpectrumHelper(std::string fname, double time_save, std::string name, double* output_spectrum, int nbins);

void InputOutput::WriteSpectrumOutput() {

    std::shared_ptr<Fields> fields_ptr = supervisor_ptr->fields_ptr;
    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;
    std::shared_ptr<TimeStepping> timestep_ptr = supervisor_ptr->timestep_ptr;

    NVTX3_FUNC_RANGE();

    if (param_ptr->debug > 0) {
        std::printf("Writing spectrum output... \n");
    }


    double t0        = param_ptr->t_initial;
    double time_save = timestep_ptr->current_time;
    double tend     = param_ptr->t_final;
    double output_spectrum[nbins];

    char data_output_name[16];
    std::sprintf(data_output_name,"spectrum.spooky");
    std::string fname = param_ptr->output_dir + std::string("/data/") + std::string(data_output_name);


    /**
     * First the energies
     *
     */

    if (param_ptr->incompressible) {

        computeSpectrum1d(fields_ptr->d_farray[vars.VX],
                        fields_ptr->d_farray[vars.VX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Kx", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VY],
                        fields_ptr->d_farray[vars.VY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Ky", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VZ],
                        fields_ptr->d_farray[vars.VZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Kz", output_spectrum, nbins);

    }

    if (param_ptr->mhd) {

        computeSpectrum1d(fields_ptr->d_farray[vars.BX],
                        fields_ptr->d_farray[vars.BX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Mx", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BY],
                        fields_ptr->d_farray[vars.BY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "My", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BZ],
                        fields_ptr->d_farray[vars.BZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Mz", output_spectrum, nbins);

    }

    if (param_ptr->boussinesq or param_ptr->heat_equation) {

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.TH],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Eth", output_spectrum, nbins);

    }

    /**
     * Then the Reynolds/Maxwell/Buoyancy spectra
     *
     */

    if (param_ptr->incompressible) {

        computeSpectrum1d(fields_ptr->d_farray[vars.VX],
                        fields_ptr->d_farray[vars.VY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "vxvy", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VY],
                        fields_ptr->d_farray[vars.VZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "vyvz", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VZ],
                        fields_ptr->d_farray[vars.VX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "vzvx", output_spectrum, nbins);

    }

    if (param_ptr->mhd) {

        computeSpectrum1d(fields_ptr->d_farray[vars.BX],
                        fields_ptr->d_farray[vars.BY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "bxby", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BY],
                        fields_ptr->d_farray[vars.BZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "bybz", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BZ],
                        fields_ptr->d_farray[vars.BX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "bzbx", output_spectrum, nbins);

    }

    if (param_ptr->boussinesq) {

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.VX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "thvx", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.VY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "thvy", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.VZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "thvz", output_spectrum, nbins);

    }



    

}


void writeSpectrumHelper(std::string fname, double time_save, std::string name, double* output_spectrum, int nbins){

    std::ofstream outputfile;
    outputfile.open (fname, std::ios_base::app);

    outputfile << "t" << "\t";
    outputfile << std::scientific << std::setprecision(8) << time_save << "\t";

    outputfile << name << "\t";

    for (int i = 0; i < nbins; i++) {
        outputfile << std::scientific << std::setprecision(8) << output_spectrum[i] << "\t";
    }

    outputfile << "\n";

    outputfile.close();


}

void InputOutput::computeSpectrum1d(data_type* v1, data_type* v2,
                       double* output_spectrum) {

    std::shared_ptr<Fields> fields_ptr = supervisor_ptr->fields_ptr;
    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;
    int nbins = fields_ptr->wavevector.nbins;
    double deltak = fields_ptr->wavevector.deltak;

    scalar_type* kvec = fields_ptr->wavevector.d_all_kvec;

    int blocksPerGrid = ( nbins + threadsPerBlock - 1) / threadsPerBlock;
    VecInit<<<blocksPerGrid, threadsPerBlock>>>(d_output_spectrum, 0.0, nbins);


    blocksPerGrid = ( grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
    Spectrum1d<<<blocksPerGrid, threadsPerBlock>>>(kvec, v1, v2, d_output_spectrum, nbins, deltak, grid.NX, grid.NY, grid.NZ, (size_t) grid.NTOTAL_COMPLEX);

    CUDA_RT_CALL(hipMemcpy(output_spectrum, d_output_spectrum, sizeof(scalar_type) * nbins, hipMemcpyDeviceToHost));


}



void InputOutput::WriteSpectrumOutputHeader() {

    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;

    int nbins = supervisor_ptr->fields_ptr->wavevector.nbins;
    double deltak = supervisor_ptr->fields_ptr->wavevector.deltak;

    if (param_ptr->debug > 0) {
        std::printf("Writing spectrum header... \n");
    }

    char data_output_name[16];
    std::sprintf(data_output_name,"spectrum.spooky");
    std::string fname = param_ptr->output_dir + std::string("/data/") + std::string(data_output_name);

    std::ofstream outputfile;
    outputfile.open (fname, std::ios_base::app);


    outputfile << "## This file contains the 1d (shell-integrated) energy spectral densities of the following quantities: \n";
    outputfile << "## \t";

    // for(int i = 0 ; i < spookyOutSpectrum.size() ; i++) {
    //     outputfile << spookyOutSpectrum[i]  << "\t";
    // }

    outputfile << "## The wavevector: \n";

    for(int i = 0 ; i < nbins ; i++) {
        outputfile << std::scientific << std::setprecision(8) << i*deltak << "\t";
    }


    outputfile << "\n";
    outputfile.close();
}
