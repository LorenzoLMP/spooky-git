#include "common.hpp"
#include "cufft_routines.hpp"
#include "cuda_kernels_generic.hpp"
#include "cuda_kernels.hpp"
// #include "../libs/HighFive/include/highfive/highfive.hpp"
// #include <highfive/highfive.hpp>
#include "fields.hpp"
// #include "hdf5_io.hpp"
// #include "output_timevar.hpp"
// #include "spooky_outputs.hpp"
#include <iostream>
#include <fstream>
#include <iomanip>
#include "parameters.hpp"
#include "inputoutput.hpp"
#include "timestepping.hpp"
#include "supervisor.hpp"

void writeSpectrumHelper(std::string fname, double time_save, std::string name, double* output_spectrum, int nbins);

void InputOutput::WriteSpectrumOutput() {

    std::shared_ptr<Fields> fields_ptr = supervisor_ptr->fields_ptr;
    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;
    std::shared_ptr<TimeStepping> timestep_ptr = supervisor_ptr->timestep_ptr;

    NVTX3_FUNC_RANGE();

    if (param_ptr->debug > 0) {
        std::printf("Writing spectrum output... \n");
    }


    double t0        = param_ptr->t_initial;
    double time_save = timestep_ptr->current_time;
    double tend     = param_ptr->t_final;
    double output_spectrum[nbins];

    char data_output_name[16];
    std::sprintf(data_output_name,"spectrum.spooky");
    std::string fname = param_ptr->output_dir + std::string("/data/") + std::string(data_output_name);

    int blocksPerGrid;

    // we already computed the c2r FFTs when the WriteTimevarOutput()
    // was called, so the real fields are saved in the d_all_buffer_r
    scalar_type* real_velField = fields_ptr->d_all_buffer_r + vars.VEL * 2 * grid.NTOTAL_COMPLEX ;
    scalar_type* real_magField = fields_ptr->d_all_buffer_r + vars.MAG * 2 * grid.NTOTAL_COMPLEX ;
    scalar_type* kvec = fields_ptr->wavevector.d_all_kvec;
    scalar_type* mask = fields_ptr->wavevector.d_mask;

    /**
     * First the energies
     *
     */

    if (param_ptr->incompressible) {

        computeSpectrum1d(fields_ptr->d_farray[vars.VX],
                        fields_ptr->d_farray[vars.VX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Kx", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VY],
                        fields_ptr->d_farray[vars.VY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Ky", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VZ],
                        fields_ptr->d_farray[vars.VZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Kz", output_spectrum, nbins);

    }

    if (param_ptr->mhd) {

        computeSpectrum1d(fields_ptr->d_farray[vars.BX],
                        fields_ptr->d_farray[vars.BX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Mx", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BY],
                        fields_ptr->d_farray[vars.BY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "My", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BZ],
                        fields_ptr->d_farray[vars.BZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Mz", output_spectrum, nbins);

    }

    if (param_ptr->boussinesq or param_ptr->heat_equation) {

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.TH],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Eth", output_spectrum, nbins);

    }

    /**
     * Then the Reynolds/Maxwell/Buoyancy spectra
     *
     */

    if (param_ptr->incompressible) {

        computeSpectrum1d(fields_ptr->d_farray[vars.VX],
                        fields_ptr->d_farray[vars.VY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "vxvy", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VY],
                        fields_ptr->d_farray[vars.VZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "vyvz", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VZ],
                        fields_ptr->d_farray[vars.VX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "vzvx", output_spectrum, nbins);

    }

    if (param_ptr->mhd) {

        computeSpectrum1d(fields_ptr->d_farray[vars.BX],
                        fields_ptr->d_farray[vars.BY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "bxby", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BY],
                        fields_ptr->d_farray[vars.BZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "bybz", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BZ],
                        fields_ptr->d_farray[vars.BX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "bzbx", output_spectrum, nbins);

    }

    if (param_ptr->boussinesq) {

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.VX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "thvx", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.VY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "thvy", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.VZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "thvz", output_spectrum, nbins);

    }

    /**
     * Then the emf spectra
     *
     */

    if (param_ptr->mhd) {

        // compute emf = u x B:
        // emf_x = u_y B_z - u_z B_y , emf_y = u_z B_x - u_x B_z , emf_z = u_x B_y - u_y B_x
        // the results are saved in the first 3 temp_arrays as [emf_x, emf_y, emf_z] (they are the x,y,z components of the emf)
        // We can re-utilize tmparrays and store result in in the temp_arrays from [0, 1, 2]

        data_type* emf = fields_ptr->d_all_tmparray;
        data_type* curlemf = fields_ptr->d_all_tmparray;

        blocksPerGrid = ( 2 * grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
        MagneticEmf<<<blocksPerGrid, threadsPerBlock>>>(real_velField, real_magField, (scalar_type*) emf,  2 * grid.NTOTAL_COMPLEX);

        // take fourier transforms of the 3 independent components of the antisymmetric shear matrix
        for (int n = 0; n < 3; n++) {
            r2c_fft((scalar_type*) emf + 2*n*grid.NTOTAL_COMPLEX, emf + n*grid.NTOTAL_COMPLEX, supervisor_ptr);
        }

        // compute curl of emf and assign to curlemf (we can re-utilize the temp arrays)
        blocksPerGrid = ( grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
        MagneticShear<<<blocksPerGrid, threadsPerBlock>>>(kvec, emf, curlemf, mask, grid.NTOTAL_COMPLEX);

        // now compute the spectrum for the 3 components
        computeSpectrum1d(fields_ptr->d_farray[vars.BX],
                        curlemf,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "emfwork_x", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BY],
                        curlemf + grid.NTOTAL_COMPLEX,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "emfwork_y", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BZ],
                        curlemf + 2*grid.NTOTAL_COMPLEX,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "emfwork_z", output_spectrum, nbins);

    }



    

}


void writeSpectrumHelper(std::string fname, double time_save, std::string name, double* output_spectrum, int nbins){

    std::ofstream outputfile;
    outputfile.open (fname, std::ios_base::app);

    outputfile << "t" << "\t";
    outputfile << std::scientific << std::setprecision(8) << time_save << "\t";

    outputfile << name << "\t";

    for (int i = 0; i < nbins; i++) {
        outputfile << std::scientific << std::setprecision(8) << output_spectrum[i] << "\t";
    }

    outputfile << "\n";

    outputfile.close();


}

void InputOutput::computeSpectrum1d(data_type* v1, data_type* v2,
                       double* output_spectrum) {

    std::shared_ptr<Fields> fields_ptr = supervisor_ptr->fields_ptr;
    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;
    int nbins = fields_ptr->wavevector.nbins;
    double deltak = fields_ptr->wavevector.deltak;

    scalar_type* kvec = fields_ptr->wavevector.d_all_kvec;

    int blocksPerGrid = ( nbins + threadsPerBlock - 1) / threadsPerBlock;
    VecInit<<<blocksPerGrid, threadsPerBlock>>>(d_output_spectrum, 0.0, nbins);


    blocksPerGrid = ( grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
    Spectrum1d<<<blocksPerGrid, threadsPerBlock>>>(kvec, v1, v2, d_output_spectrum, nbins, deltak, grid.NX, grid.NY, grid.NZ, (size_t) grid.NTOTAL_COMPLEX);

    CUDA_RT_CALL(hipMemcpy(output_spectrum, d_output_spectrum, sizeof(scalar_type) * nbins, hipMemcpyDeviceToHost));


}



void InputOutput::WriteSpectrumOutputHeader() {

    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;

    int nbins = supervisor_ptr->fields_ptr->wavevector.nbins;
    double deltak = supervisor_ptr->fields_ptr->wavevector.deltak;

    if (param_ptr->debug > 0) {
        std::printf("Writing spectrum header... \n");
    }

    char data_output_name[16];
    std::sprintf(data_output_name,"spectrum.spooky");
    std::string fname = param_ptr->output_dir + std::string("/data/") + std::string(data_output_name);

    std::ofstream outputfile;
    outputfile.open (fname, std::ios_base::app);


    outputfile << "## This file contains the 1d (shell-integrated) energy spectral densities of the following quantities: \n";
    outputfile << "## \t";

    // for(int i = 0 ; i < spookyOutSpectrum.size() ; i++) {
    //     outputfile << spookyOutSpectrum[i]  << "\t";
    // }

    outputfile << "## The wavevector: \n";

    for(int i = 0 ; i < nbins ; i++) {
        outputfile << std::scientific << std::setprecision(8) << i*deltak << "\t";
    }


    outputfile << "\n";
    outputfile.close();
}
