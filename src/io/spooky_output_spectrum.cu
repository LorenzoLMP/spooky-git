#include "common.hpp"
#include "cufft_routines.hpp"
#include "cuda_kernels_generic.hpp"
#include "cuda_kernels.hpp"
// #include "../libs/HighFive/include/highfive/highfive.hpp"
// #include <highfive/highfive.hpp>
#include "fields.hpp"
// #include "hdf5_io.hpp"
// #include "output_timevar.hpp"
// #include "spooky_outputs.hpp"
#include <iostream>
#include <fstream>
#include <iomanip>
#include "parameters.hpp"
#include "inputoutput.hpp"
#include "timestepping.hpp"
#include "supervisor.hpp"
#include "physics.hpp"

void writeSpectrumHelper(std::string fname, double time_save, std::string name, double* output_spectrum, int nbins);

void InputOutput::WriteSpectrumOutput() {

    std::shared_ptr<Fields> fields_ptr = supervisor_ptr->fields_ptr;
    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;
    std::shared_ptr<TimeStepping> timestep_ptr = supervisor_ptr->timestep_ptr;

    NVTX3_FUNC_RANGE();

    if (param_ptr->debug > 0) {
        std::printf("Writing spectrum output... \n");
    }


    double t0        = param_ptr->t_initial;
    double time_save = timestep_ptr->current_time;
    double tend     = param_ptr->t_final;
    double output_spectrum[nbins];

    char data_output_name[16];
    std::sprintf(data_output_name,"spectrum.spooky");
    std::string fname = param_ptr->output_dir + std::string("/data/") + std::string(data_output_name);

    /**
     * First the energies
     *
     */

    if (param_ptr->incompressible) {

        computeSpectrum1d(fields_ptr->d_farray[vars.VX],
                        fields_ptr->d_farray[vars.VX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Kx", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VY],
                        fields_ptr->d_farray[vars.VY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Ky", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VZ],
                        fields_ptr->d_farray[vars.VZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Kz", output_spectrum, nbins);

    }

    if (param_ptr->mhd) {

        computeSpectrum1d(fields_ptr->d_farray[vars.BX],
                        fields_ptr->d_farray[vars.BX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Mx", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BY],
                        fields_ptr->d_farray[vars.BY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "My", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BZ],
                        fields_ptr->d_farray[vars.BZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Mz", output_spectrum, nbins);

    }

    if (param_ptr->boussinesq or param_ptr->heat_equation) {

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.TH],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "Eth", output_spectrum, nbins);

    }

    /**
     * Then the Reynolds/Maxwell/Buoyancy spectra
     *
     */

    if (param_ptr->incompressible) {

        computeSpectrum1d(fields_ptr->d_farray[vars.VX],
                        fields_ptr->d_farray[vars.VY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "vxvy", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VY],
                        fields_ptr->d_farray[vars.VZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "vyvz", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VZ],
                        fields_ptr->d_farray[vars.VX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "vzvx", output_spectrum, nbins);

    }

    if (param_ptr->mhd) {

        computeSpectrum1d(fields_ptr->d_farray[vars.BX],
                        fields_ptr->d_farray[vars.BY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "bxby", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BY],
                        fields_ptr->d_farray[vars.BZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "bybz", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BZ],
                        fields_ptr->d_farray[vars.BX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "bzbx", output_spectrum, nbins);

    }

    if (param_ptr->boussinesq) {

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.VX],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "thvx", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.VY],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "thvy", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        fields_ptr->d_farray[vars.VZ],
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "thvz", output_spectrum, nbins);

    }


    // // we already computed the c2r FFTs when the WriteTimevarOutput()
    // // was called, so the real fields are saved in the d_all_buffer_r
    
    /**
     * Now the nonlinear advection term (u \cdot \nabla) u
     *
     */

    if (param_ptr->incompressible) {

        scalar_type* real_velField = fields_ptr->d_all_buffer_r + vars.VEL * 2 * grid.NTOTAL_COMPLEX ;

        data_type* divshear = fields_ptr->d_all_tmparray;

        supervisor_ptr->phys_ptr->NonLinearAdvection(real_velField, divshear);
        
        // now compute the nonlinear advection spectrum with the 3 components of divshear
        computeSpectrum1d(fields_ptr->d_farray[vars.VX],
                        divshear,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "advec_x", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VY],
                        divshear + grid.NTOTAL_COMPLEX,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "advec_y", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VZ],
                        divshear + 2*grid.NTOTAL_COMPLEX,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "advec_z", output_spectrum, nbins);

    }

    /**
     * Then the emf spectra
     *
     */

    if (param_ptr->mhd) {

        data_type* curlemf = fields_ptr->d_all_tmparray;
        supervisor_ptr->phys_ptr->CurlEMF(fields_ptr->d_all_fields, fields_ptr->d_all_buffer_r, curlemf);


        // now compute the spectrum for the 3 components
        computeSpectrum1d(fields_ptr->d_farray[vars.BX],
                        curlemf,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "emfpower_x", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BY],
                        curlemf + grid.NTOTAL_COMPLEX,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "emfpower_y", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BZ],
                        curlemf + 2*grid.NTOTAL_COMPLEX,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "emfpower_z", output_spectrum, nbins);

    }

    /**
     * Then the Lorentz transfer spectra
     *
     */

    if (param_ptr->mhd) {

        scalar_type* real_magField = fields_ptr->d_all_buffer_r + vars.MAG * 2 * grid.NTOTAL_COMPLEX ;

        data_type* divshear = fields_ptr->d_all_tmparray;

        supervisor_ptr->phys_ptr->NonLinearAdvection(real_magField, divshear);

        // now compute the Lorentz spectrum with the 3 components of divshear
        computeSpectrum1d(fields_ptr->d_farray[vars.VX],
                        divshear,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "lorentz_x", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VY],
                        divshear + grid.NTOTAL_COMPLEX,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "lorentz_y", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.VZ],
                        divshear + 2*grid.NTOTAL_COMPLEX,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "lorentz_z", output_spectrum, nbins);


    }

    /**
     * Then the nonlinear thermal advection 
     *
     */

    if (param_ptr->boussinesq) {

        data_type* thermal_adv = fields_ptr->d_all_tmparray;

        supervisor_ptr->phys_ptr->AdvectTemperature(fields_ptr->d_all_fields, fields_ptr->d_all_buffer_r, thermal_adv);

        // now compute the thermal nonlinear adv spectrum with thermal_adv
        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        thermal_adv,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "th_advec", output_spectrum, nbins);

    }


    /**
     * These are the helicity spectra
     * 
     */

    if (param_ptr->mhd) {

        data_type* complex_dMag = fields_ptr->d_all_fields + vars.MAG * grid.NTOTAL_COMPLEX ;
        data_type* mag_helicity = fields_ptr->d_all_tmparray;

        int blocksPerGrid;
        scalar_type* kvec = fields_ptr->wavevector.d_all_kvec;

        blocksPerGrid = ( grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;

        Helicity<<<blocksPerGrid, threadsPerBlock>>>(kvec, complex_dMag, mag_helicity, grid.NTOTAL_COMPLEX);

        // now compute the Helicity spectra with the 3 components of mag_helicity
        computeSpectrum1d(fields_ptr->d_farray[vars.BX],
                        mag_helicity,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "helicity_x", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BY],
                        mag_helicity + grid.NTOTAL_COMPLEX,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "helicity_y", output_spectrum, nbins);

        computeSpectrum1d(fields_ptr->d_farray[vars.BZ],
                        mag_helicity + 2*grid.NTOTAL_COMPLEX,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "helicity_z", output_spectrum, nbins);
        
    }



    /**
     * These are the anisotropic conduction terms
     * 
     */

    // first the dissipation
    if (param_ptr->anisotropic_diffusion) {

        // this is the destination temp array for the anisotropic
        // dissipation
        data_type* aniso_dissipation = fields_ptr->d_tmparray[4];
        
        supervisor_ptr->phys_ptr->AnisotropicDissipation(fields_ptr->d_all_fields, 
                                                        fields_ptr->d_all_buffer_r, 
                                                        aniso_dissipation);
        
        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        aniso_dissipation,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "th_aniso_diss", output_spectrum, nbins);

    }

    // then the anisotropic injection
    if (param_ptr->anisotropic_diffusion) {

        // this is the destination temp array for the anisotropic
        // injection
        data_type* aniso_injection = fields_ptr->d_tmparray[4];
        
        supervisor_ptr->phys_ptr->AnisotropicInjection(fields_ptr->d_all_fields, 
                                                        fields_ptr->d_all_buffer_r, 
                                                        aniso_injection);
        
        computeSpectrum1d(fields_ptr->d_farray[vars.TH],
                        aniso_injection,
                        output_spectrum);
        writeSpectrumHelper(fname, time_save, "th_aniso_inj", output_spectrum, nbins);

    }


}


void writeSpectrumHelper(std::string fname, double time_save, std::string name, double* output_spectrum, int nbins){

    std::ofstream outputfile;
    outputfile.open (fname, std::ios_base::app);

    outputfile << "t" << "\t";
    outputfile << std::scientific << std::setprecision(8) << time_save << "\t";

    outputfile << name << "\t";

    for (int i = 0; i < nbins; i++) {
        outputfile << std::scientific << std::setprecision(8) << output_spectrum[i] << "\t";
    }

    outputfile << "\n";

    outputfile.close();


}

void InputOutput::computeSpectrum1d(data_type* v1, data_type* v2,
                       double* output_spectrum) {

    std::shared_ptr<Fields> fields_ptr = supervisor_ptr->fields_ptr;
    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;
    int nbins = fields_ptr->wavevector.nbins;
    double deltak = fields_ptr->wavevector.deltak;

    scalar_type* kvec = fields_ptr->wavevector.d_all_kvec;

    int blocksPerGrid = ( nbins + threadsPerBlock - 1) / threadsPerBlock;
    VecInit<<<blocksPerGrid, threadsPerBlock>>>(d_output_spectrum, 0.0, nbins);


    blocksPerGrid = ( grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
    Spectrum1d<<<blocksPerGrid, threadsPerBlock>>>(kvec, v1, v2, d_output_spectrum, nbins, deltak, grid.NX, grid.NY, grid.NZ, (size_t) grid.NTOTAL_COMPLEX);

    CUDA_RT_CALL(hipMemcpy(output_spectrum, d_output_spectrum, sizeof(scalar_type) * nbins, hipMemcpyDeviceToHost));


}



void InputOutput::WriteSpectrumOutputHeader() {

    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;

    int nbins = supervisor_ptr->fields_ptr->wavevector.nbins;
    double deltak = supervisor_ptr->fields_ptr->wavevector.deltak;

    if (param_ptr->debug > 0) {
        std::printf("Writing spectrum header... \n");
    }

    char data_output_name[16];
    std::sprintf(data_output_name,"spectrum.spooky");
    std::string fname = param_ptr->output_dir + std::string("/data/") + std::string(data_output_name);

    std::ofstream outputfile;
    outputfile.open (fname, std::ios_base::app);


    outputfile << "## This file contains the 1d (shell-integrated) energy spectral densities of the following quantities: \n";
    outputfile << "## \t";

    // for(int i = 0 ; i < spookyOutSpectrum.size() ; i++) {
    //     outputfile << spookyOutSpectrum[i]  << "\t";
    // }

    outputfile << "## The wavevector: \n";

    for(int i = 0 ; i < nbins ; i++) {
        outputfile << std::scientific << std::setprecision(8) << i*deltak << "\t";
    }


    outputfile << "\n";
    outputfile.close();
}
