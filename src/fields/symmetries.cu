#include "define_types.hpp"
#include "fields.hpp"
#include "cufft_routines.hpp"
#include "spooky.hpp"
#include "common.hpp"
#include "cublas_routines.hpp"
#include "cuda_kernels.hpp"
#include "parameters.hpp"
#include "timestepping.hpp"
#include "supervisor.hpp"

void Fields::CheckSymmetries(){

    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;

    int current_step = supervisor_ptr->timestep_ptr->current_step;
    int symmetries_step = param_ptr->symmetries_step;
    double deltax = param_ptr->lx / grid.NX;
    double meanFieldDiv = 0.0;


    if (param_ptr->debug > 0 and current_step % 100 == 0) {
        std::printf("Computing divergence of v/B fields \n");
        if (param_ptr->incompressible) {
            // compute mean divergence for velocity field
            meanFieldDiv = ComputeDivergence(d_all_fields + grid.NTOTAL_COMPLEX * vars.VEL);
            std::printf("---- Mean-divergence of v-field is %.2e [< div Field> * Delta x]\n", meanFieldDiv*deltax);
        }
        if (param_ptr->mhd) {
            meanFieldDiv = ComputeDivergence(d_all_fields + grid.NTOTAL_COMPLEX * vars.MAG);
            std::printf("---- Mean-divergence of B-field is %.2e [< div Field> * Delta x]\n", meanFieldDiv*deltax);
        }
    }



    if( current_step % symmetries_step) {
        CleanFieldDivergence();


//         // clean divergence for velocity field
//         CleanFieldDivergence(d_all_fields + grid.NTOTAL_COMPLEX * vars.VEL);

//         // clean divergence for magnetic field
//         CleanFieldDivergence(d_all_fields + grid.NTOTAL_COMPLEX * vars.MAG);

    }
    
}

double Fields::ComputeDivergence( data_type* complex_Fields ){

    double FieldDiv = 0.0;
    // double divBfield = 0.0;
    int blocksPerGrid;
    hipblasStatus_t stat;


    blocksPerGrid = ( grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
    Divergence<<<blocksPerGrid, threadsPerBlock>>>(wavevector.d_all_kvec, complex_Fields, (data_type *)  d_tmparray[0], (size_t) grid.NTOTAL_COMPLEX);
    // transform back to real space
    c2r_fft(d_tmparray[0], d_tmparray_r[0]);
    // compute absolute value of real vector (actually Dasum already does it...)
    // reduce sum
    stat = hipblasDasum(handle0, 2 * grid.NTOTAL_COMPLEX, d_tmparray_r[0], 1, &FieldDiv);
    if (stat != HIPBLAS_STATUS_SUCCESS) std::printf("- Reduce-sum of ComputeDivergence failed\n");
    // std::printf("----Mean-divergence of v-field is %.2e / L\n",divvfield/(2 * grid.NTOTAL_COMPLEX));

    return FieldDiv/(2 * grid.NTOTAL_COMPLEX);

}

// void Fields::CleanDivergence(){
//

//         // clean divergence for velocity field
//         CleanFieldDivergence(d_all_fields + grid.NTOTAL_COMPLEX * vars.VEL);

//         // clean divergence for magnetic field
//         CleanFieldDivergence(d_all_fields + grid.NTOTAL_COMPLEX * vars.MAG);

//
// }

void Fields::CleanFieldDivergence( ){

    int blocksPerGrid;

    blocksPerGrid = ( grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;


    if (param_ptr->incompressible) {
        CleanDivergence<<<blocksPerGrid, threadsPerBlock>>>(wavevector.d_all_kvec, d_all_fields + grid.NTOTAL_COMPLEX * vars.VEL, d_all_fields + grid.NTOTAL_COMPLEX * vars.VEL, (size_t) grid.NTOTAL_COMPLEX);
    }
    if (param_ptr->mhd) {
        CleanDivergence<<<blocksPerGrid, threadsPerBlock>>>(wavevector.d_all_kvec, d_all_fields + grid.NTOTAL_COMPLEX * vars.MAG, d_all_fields + grid.NTOTAL_COMPLEX * vars.MAG, (size_t) grid.NTOTAL_COMPLEX);
    }

}
