#include "hip/hip_runtime.h"
#include "common.hpp"
// #include "wavevector.hpp"
#include "cufft_routines.hpp"
#include "fields.hpp"
#include "parameters.hpp"
#include "cuda_kernels_generic.hpp"
#include "supervisor.hpp"

Wavevector::~Wavevector() {
    // free(kxt);
    // free(ky);
    // free(kz);
    free(all_kvec);
    free(kvec);
    free(mask);

    free(d_kvec);
}


// void Wavevector::init_Wavevector(Parameters *p_in) {
Wavevector::Wavevector(Supervisor &sup_in, Parameters &p_in) {

    supervisor_ptr = &sup_in;
    unsigned int idx;

    // scalar_type Lx, scalar_type Ly, scalar_type Lz
    lx = p_in.lx; ly = p_in.ly; lz = p_in.lz;
    // lx = Lx; ly = Ly; lz = Lz;
    // std::printf("baginning of wave\n");
    // all_kvec contains kx ky kz sequentially
    all_kvec = (scalar_type *) malloc( (size_t) sizeof(scalar_type) * grid.NTOTAL_COMPLEX * 3);
    // kvec is array of arrays such that kvec[0] = kx, etc
    kvec = (scalar_type **) malloc( (size_t) sizeof(scalar_type) * 3);
    // init kvec
    // std::printf("before init kvec\n");
    for (int i = 0 ; i < 3 ; i++) {
        kvec[i]   = all_kvec + i*grid.NTOTAL_COMPLEX;
    }

    d_kvec = (scalar_type **) malloc( (size_t) sizeof(scalar_type *) * 3);
    // kxt = (scalar_type *) malloc( (size_t) sizeof(scalar_type) * grid.NTOTAL_COMPLEX);
    // ky = (scalar_type *) madouble Lx, double Ly, double Lzlloc( (size_t) sizeof(scalar_type) * grid.NTOTAL_COMPLEX);
    // kz = (scalar_type *) malloc( (size_t) sizeof(scalar_type) * grid.NTOTAL_COMPLEX);
    // kz = (scalar_type *) malloc( (size_t) sizeof(scalar_type) * grid.NTOTAL_COMPLEX);


    mask = (scalar_type *) malloc( (size_t) sizeof(scalar_type) * grid.NTOTAL_COMPLEX);

    for (int i = 0; i < grid.NX; i++){
        for (int j = 0; j < grid.NY; j++){
            for (int k = 0; k < grid.NZ/2 + 1; k++){
                idx = k + (grid.NZ/2+1) * ( j + i * grid.NY);
                kvec[vars.KX][idx] = (2.0 * M_PI) / lx * (fmod( (double) i + ( (double) grid.NX / 2) ,  grid.NX ) - (double) grid.NX / 2 );
                kvec[vars.KY][idx]  = (2.0 * M_PI) / ly * (fmod( (double) j + ( (double) grid.NY / 2) ,  grid.NY ) - (double) grid.NY / 2 );
                kvec[vars.KZ][idx]  = (2.0 * M_PI) / lz * (double) k;
            }
        }
    }

    std::printf("Finished filling wavevector\n");


    kxmax = 2.0 * M_PI/ lx * ( ( (double) grid.NX / 2) - 1);
    kymax = 2.0 * M_PI/ ly * ( ( (double) grid.NY / 2) - 1);
    kzmax = 2.0 * M_PI/ lz * ( ( (double) grid.NZ / 2) - 1);

    std::printf("Maximum wavenumbers (without dealiasing): kxmax = %.2e  kymax = %.2e  kzmax = %.2e \n",kxmax,kymax, kzmax);

    if (p_in.antialiasing){
        for (int i = 0; i < grid.NX; i++){
            for (int j = 0; j < grid.NY; j++){
                for (int k = 0; k < grid.NZ/2 + 1; k++){
                    idx = k + (grid.NZ/2+1) * ( j + i * grid.NY);
                    mask[idx] = 1.0;
                    if( fabs( kvec[vars.KX][ idx] ) > 2.0/3.0 * kxmax)
                        mask[idx] = 0.0;
                    if( fabs( kvec[vars.KY][ idx ] ) > 2.0/3.0 * kymax)
                        mask[idx] = 0.0;
                    if( fabs( kvec[vars.KZ][ idx ] ) > 2.0/3.0 * kzmax)
                        mask[idx] = 0.0;
                }
            }
        }
        kxmax = (2.0 / 3.0 ) * kxmax;
        kymax = (2.0 / 3.0 ) * kymax;
        kzmax = (2.0 / 3.0 ) * kzmax;
    }

    kmax  = pow(kxmax*kxmax+kymax*kymax+kzmax*kzmax,0.5);
}

void Wavevector::print_values() {
    int idx;
    const char* k_comp[3] = {"kx", "ky", "kz"};
    // for (int i = 0; i < 25; i++){
    //     idx =  (grid.NZ/2+1)*2 * ( i * grid.NY);
    //     // std::printf("v1[%d]= %f \t v2[%d]= %f \n", idx, farray_r[0][idx], idx, farray_r[1][idx]);
    //     for (int n = 0; n < vars.NUM_FIELDS; n++){
    //         std::printf("v[%d][%d] = %.7e \t", n, idx, farray_r[n][idx]);
    //     }
    //     std::cout << std::endl;
    // }
    for (int i = 0; i < 2; i++){
        for (int j = 0; j < 2; j++){
            for (int k = 0; k < 2; k++){
                idx = k + (grid.NZ/2+1)*2 * ( j + i * grid.NY);
                // std::printf("v1[%d]= %f \t v2[%d]= %f \n", idx, farray_r[0][idx], idx, farray_r[1][idx]);
                for (int n = 0; n < 3; n++){
                    std::printf("k[%s][%d] = %.3e \t", k_comp[n], idx, kvec[n][idx]);
                }
                std::cout << std::endl;
            }
        }
    }
    // std::printf("kx:\n");
    // int i,j,k, idx;
    // j = 0; k = 0;
    // for (i = 0; i < 10; i++){
    //     idx = k + (grid.NZ/2+1) * ( j + i * grid.NY);
    //     std::printf("kx[%d] %f \n", i, kvec[vars.KX][idx]);
    // }
    // i = 0; k = 0;
    // std::printf("ky:\n");
    // for (j = 0; j < 10; j++){
    //     idx = k + (grid.NZ/2+1) * ( j + i * grid.NY);
    //     std::printf("ky[%d] %f \n", j, kvec[vars.KY][idx]);
    // }
    // i = 0; j = 0;
    // std::printf("kz:\n");
    // for (k = 0; k < 10; k++){
    //     idx = k + (grid.NZ/2+1) * ( j + i * grid.NY);
    //     std::printf("kz[%d] %f \n", k, kvec[vars.KZ][idx]);
    // }
}

// void Wavevector::shear_Wavevector( double t, double dt) {
//     // write routines for shearing kxt
// }

void Wavevector::shearWavevector(double tremap){

    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;
    // tremap is already non-dimensionalized qty: param.shear * tremap
    double kxmin = (2.0 * M_PI) / param_ptr->lx;

    int blocksPerGrid = (grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;

    ShearWavevector<<<blocksPerGrid, threadsPerBlock>>>( d_kvec[vars.KX], d_kvec[vars.KY], tremap*param_ptr->shear, kxmin, grid.FFT_SIZE, grid.NTOTAL_COMPLEX);

}

void Wavevector::allocate_and_move_to_gpu() {
    // void *d_vx, *d_vy;

    CUDA_RT_CALL(hipMalloc(&d_all_kvec, (size_t) sizeof(scalar_type) * grid.NTOTAL_COMPLEX * 3));

    CUDA_RT_CALL(hipMalloc(&d_mask,  (size_t) sizeof(scalar_type) * grid.NTOTAL_COMPLEX));

    CUDA_RT_CALL(hipMemcpy(d_all_kvec, all_kvec, (size_t) sizeof(scalar_type) * grid.NTOTAL_COMPLEX * 3, hipMemcpyHostToDevice));

    CUDA_RT_CALL(hipMemcpy(d_mask, mask, (size_t) sizeof(scalar_type) * grid.NTOTAL_COMPLEX, hipMemcpyHostToDevice));

    for (int i = 0 ; i < 3 ; i++) {
        d_kvec[i]  = d_all_kvec + i*grid.NTOTAL_COMPLEX;
    }

}

void Wavevector::sync_with_host() {
    CUDA_RT_CALL(hipMemcpy(kvec[vars.KX], d_kvec[vars.KX], (size_t) sizeof(scalar_type) * grid.NTOTAL_COMPLEX, hipMemcpyDeviceToHost));

}

void Wavevector::clean_gpu(){
    CUDA_RT_CALL(hipFree(d_all_kvec));
    CUDA_RT_CALL(hipFree(d_mask));
}

