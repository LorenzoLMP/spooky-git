#include "define_types.hpp"
#include "supervisor.hpp"
// #include "cufft_routines.hpp"
#include "spooky.hpp"
#include "common.hpp"
#include "cublas_routines.hpp"
#include "cuda_kernels.hpp"
#include "parameters.hpp"
// #include "inputoutput.hpp"
#include "fields.hpp"
#include <hip/hip_runtime.h>
// #include <hipfft/hipfftXt.h>
// #include "spooky.hpp"
#include "cufft_utils.h"
// #include "define_types.hpp"

Supervisor::Supervisor() {

    time_delta = 0.0;
    NumFFTs = 0; // in mainloop
    // NumFFTs[1] = 0; // in IO
    TimeSpentInFFTs = 0.0;
    TimeSpentInMainLoop = 0.0;

    AllocCpuMem = 0;
    AllocGpuMem = 0;

    ElapsedWallClockTime = 0.0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventCreate(&start_2);
    hipEventCreate(&stop_2);
}

void Supervisor::updateFFTtime(){
    // in ms
    hipEventElapsedTime(&time_delta, start, stop);
    // in s
    TimeSpentInFFTs += 1e-3*time_delta;
}

void Supervisor::updateMainLooptime(){
    // in ms
    hipEventElapsedTime(&time_delta_2, start_2, stop_2);
    // in s
    TimeSpentInMainLoop += 1e-3*time_delta_2;
}

Supervisor::~Supervisor(){

}
