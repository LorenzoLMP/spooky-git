#include "common.hpp"
#include "supervisor.hpp"
#include "parameters.hpp"
#include "fields.hpp"
#include "inputoutput.hpp"
#include "timestepping.hpp"
#include "physics.hpp"

#include "cublas_routines.hpp"
#include "cuda_kernels.hpp"
#include "cuda_kernels_generic.hpp"
#include "cufft_routines.hpp"

#include <hip/hip_runtime.h>
// #include <hipfft/hipfftXt.h>
#include "cufft_utils.h"


Supervisor::Supervisor(std::string input_dir) :
        total_timer() ,
        timevar_timer(),
        datadump_timer() {

    param_ptr = std::shared_ptr<Parameters> (new Parameters(*this, input_dir));

    /*****
     *
     * Populate Variables/Grid struct
     *
     * ****/
    param_ptr->popVariablesGrid();

    if (not param_ptr->checkParameters()){
        std::cout << "Error: your choice of physics modules is not consistent. Aborting now." << std::endl;
        exit(0);
    }


    fields_ptr = std::shared_ptr<Fields> (new Fields(*this, *param_ptr));
    phys_ptr = std::shared_ptr<Physics> (new Physics(*this));
    timestep_ptr = std::shared_ptr<TimeStepping> (new TimeStepping(*this, *param_ptr));
    inout_ptr = std::shared_ptr<InputOutput> (new InputOutput(*this));


    stats_frequency = -1;
    time_delta = 0.0;
    NumFFTs = 0; // in mainloop
    // NumFFTs[1] = 0; // in IO
    TimeSpentInFFTs = 0.0;
    TimeSpentInMainLoop = 0.0;
    TimeSpentInMainLoopPartial = 0.0;

    TimeIOTimevar = 0.0;
    TimeIODatadump = 0.0;

    AllocCpuMem = 0;
    AllocGpuMem = 0;

    ElapsedWallClockTime = 0.0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventCreate(&start_2);
    hipEventCreate(&stop_2);
}

void Supervisor::updateFFTtime(){
    // in ms
    hipEventElapsedTime(&time_delta, start, stop);
    // in s
    TimeSpentInFFTs += 1e-3*time_delta;
}

void Supervisor::updateMainLooptime(){
    // in ms
    hipEventElapsedTime(&time_delta_2, start_2, stop_2);
    // in s
    TimeSpentInMainLoop += 1e-3*time_delta_2;
    TimeSpentInMainLoopPartial += 1e-3*time_delta_2;
}

void Supervisor::print_partial_stats(){

    std::printf("---- The avg number of cell updates / sec is %.4e [cell_updates/s]  \n",grid.NTOTAL*stats_frequency/TimeSpentInMainLoopPartial);

    TimeSpentInMainLoopPartial = 0.0;
}

void Supervisor::print_final_stats(){

    int tot_steps = timestep_ptr->current_step;
    std::printf("@@@@@ ------------------------------------------------------------------ @@@@@ \n");
    std::printf("@@ ------------------------------------------------------------------------ @@ \n");
    std::printf("@@\t \t \t FINAL STATISTICS REPORT \n");
    std::printf("@@\tThe total execution time was: \t\t\t %.2e [s]  \n", total_timer.elapsed());
    std::printf("@@\tThe time spent in the mainloop was: \t\t %.2e [s]  \n",TimeSpentInMainLoop);
    std::printf("@@\tThe time spent in FFTs in the mainloop was: \t %.2e [s]  \n", TimeSpentInFFTs);
    std::printf("@@\tThe time spent in IO (timevar+datadumps): \t %.2e [s]  \n", TimeIOTimevar+TimeIODatadump);
    std::printf("@@\t\t- time spent in timevar: \t\t %.2e [s]  \n", TimeIOTimevar);
    std::printf("@@\t\t- time spent in datadumps: \t\t %.2e [s]  \n",TimeIODatadump);

    std::cout << "@@" << std::endl;
    std::printf("@@\tThe mainloop took %d FFTs and %d steps to complete \n", NumFFTs, tot_steps);
    std::printf("@@\t\t- FFTs per loop: %d \n", NumFFTs/tot_steps);
    std::cout << "@@" << std::endl;
    std::printf("@@\tThe average performance is: \n@@\t\t\t %.4e [cell_updates/s]  \n",grid.NTOTAL*tot_steps/TimeSpentInMainLoop);

    std::printf("@@ ------------------------------------------------------------------------ @@ \n");
    std::printf("@@@@@ ------------------------------------------------------------------ @@@@@ \n");
}

void Supervisor::displayConfiguration(){

    std::printf("You are running SPOOKY with the following configuration: \n");
    std::printf("Number of fields = %d \n",vars.NUM_FIELDS);
    std::printf("Grid size (real space) (NX, NY, NZ) = (%d \t %d \t %d)\n",grid.NX, grid.NY, grid.NZ);
    std::printf("Size of the domain: lx = %f \t ly = %f \t lz = %f\n",param_ptr->lx, param_ptr->ly, param_ptr->lz);
    std::printf("kxmax = %.4e  kymax = %.4e  kzmax = %.4e \n",fields_ptr->wavevector.kxmax,fields_ptr->wavevector.kymax, fields_ptr->wavevector.kzmax);

    if (param_ptr->incompressible) {
        std::printf("Re = %.4e \n",param_ptr->reynolds);
    }
    if (param_ptr->boussinesq or param_ptr->heat_equation) {
        std::printf("Re_th = %.4e \n",param_ptr->reynolds_th);
    }
    if (param_ptr->mhd) {
        std::printf("Re_m = %.4e \n",param_ptr->reynolds_m);
    }
    if (param_ptr->anisotropic_diffusion) {
        std::printf("Re_aniso = %.4e \n",param_ptr->reynolds_ani);
        std::printf("Omega_T2 = %.4e \n",param_ptr->OmegaT2);
    }

    if (param_ptr->stratification) {
        std::printf("Background stratification in direction: %d \n",param_ptr->strat_direction);
        std::printf("N2 = %.4e \n",param_ptr->N2);

    }
    if (param_ptr->shearing) {
        std::printf("S = %.4e \n",param_ptr->shear);
    }
    if (param_ptr->rotating) {
        std::printf("Omega = %.4e \n",param_ptr->omega);
    }

    if (param_ptr->supertimestepping) {
        // std::printf("Algorithm for supertimestepping: %s \n",param_ptr->sts_algorithm);
        std::cout << "Algorithm for supertimestepping: " << param_ptr->sts_algorithm << std::endl;
    }

    std::printf("t_initial = %.4e \n",param_ptr->t_initial);
    std::printf("t_current = %.4e \n",timestep_ptr->current_time);
    std::printf("t_final = %.4e \n",param_ptr->t_final);

    std::printf("Enforcing symmetries every %d steps \n",param_ptr->symmetries_step);
    std::printf("Saving snapshot every  dt = %.2e \n",param_ptr->toutput_flow);
    std::printf("Saving timevar every  dt = %.2e \n",param_ptr->toutput_time);
    std::printf("Displaying stats every num steps = %d \n",stats_frequency);
    std::printf("Maximum wallclock elapsed time (in hours): %.4e \n", param_ptr->max_walltime_elapsed);

}

void Supervisor::executeMainLoop(){

    if (param_ptr->shearing) {
        timestep_ptr->ShiftTime();
        fields_ptr->wavevector.shearWavevector(timestep_ptr->tremap);
        std::printf("t_remap = %.4e \n",timestep_ptr->tremap);
    }

    // TimeSpentInMainLoop is in seconds while
    // max_walltime_elapsed is in hours
    while (timestep_ptr->current_time < param_ptr->t_final and TimeSpentInMainLoop < param_ptr->max_walltime_elapsed*3600) {

        // advance the equations (field(n+1) = field(n) + dfield*dt)
        // timestep_ptr->RungeKutta3();
        timestep_ptr->HydroMHDAdvance(fields_ptr);
        // check if we need to output data
        inout_ptr->CheckOutput();
        // check if we need to enforce symmetries
        fields_ptr->CheckSymmetries();

        if (param_ptr->debug == 2){
            std::printf("step: %d \t dt: %.2e \n", timestep_ptr->current_step,timestep_ptr->current_dt);
        }

        if (stats_frequency > 0){
            if ( timestep_ptr->current_step % stats_frequency == 0)
            print_partial_stats();
        }

        if (TimeSpentInMainLoop >= param_ptr->max_walltime_elapsed*3600){
            std::printf("The maximum wallclock elapsed time was reached. Stopping now. You can later resume from the last snapshot available. \n");
        }


    }
}

void Supervisor::initialDataDump(){

    if (param_ptr->restart == 0){

        std::printf("Initial data dump...\n");
        try {
        inout_ptr->CheckOutput();
        }
        catch (const std::exception& err) {
        std::cerr << err.what() << std::endl;
        std::exit(1);
        }
    }
}


void Supervisor::Restart(int restart_num){
    if (param_ptr->restart == 1){
        inout_ptr->ReadDataFile(restart_num);
    }
}

Supervisor::~Supervisor(){

}

void Supervisor::Complex2RealFields(data_type* ComplexField_in, int num_fields){

    // version with in-place transform
    // compute FFTs from complex to real fields
    for (int n = 0; n < num_fields; n++){
        c2r_fft(ComplexField_in + n * grid.NTOTAL_COMPLEX,  ((scalar_type*) ComplexField_in) + n * 2*grid.NTOTAL_COMPLEX, this);
    }

}

void Supervisor::Complex2RealFields(data_type* ComplexField_in, scalar_type* RealField_out, int num_fields){

    // assign fields to [num_fields] tmparray (memory block starts at d_all_tmparray)
    int blocksPerGrid = ( num_fields * grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
    ComplexVecAssign<<<blocksPerGrid, threadsPerBlock>>>(ComplexField_in, (data_type*) RealField_out, num_fields * grid.NTOTAL_COMPLEX);

    // compute FFTs from complex to real fields
    for (int n = 0; n < num_fields; n++){
        c2r_fft((data_type*) RealField_out + n * grid.NTOTAL_COMPLEX,  RealField_out + n * 2*grid.NTOTAL_COMPLEX, this);
    }

}
