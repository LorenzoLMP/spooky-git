#include <hipblas.h>
#include "cublas_routines.hpp"

hipblasHandle_t handle0;
hipblasHandle_t handle1;

void init_cublas(){
    hipblasCreate(&handle0);
    hipblasCreate(&handle1);
}

void finish_cublas(){
    hipblasDestroy(handle0);
    hipblasDestroy(handle1);
}
