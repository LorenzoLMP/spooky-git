#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
// #include "spooky.hpp"
#include "cufft_utils.h"
#include "define_types.hpp"
// #include "cufft_routines.hpp"
// #include "common.hpp"

__global__ void scaleKernel(hipfftDoubleComplex *ft, scalar_type scale, size_t N) {
    size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        ft[i].x *= scale;
        ft[i].y *= scale;
    }
}

__global__ void RRvectorMultiply(const scalar_type *A, const scalar_type *B, scalar_type *C, scalar_type a, size_t N) {
    size_t i = i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        C[i] = a * A[i] * B[i];
    }
}

// __global__ void ComplexVecAssign(const hipfftDoubleComplex *A, hipfftDoubleComplex *B, size_t N) {
//     size_t i = i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
//
//     if (i < N) {
//         B[i].x = A[i].x;
//         B[i].y = A[i].y;
//     }
// }

__global__ void ComplexVecAssign(const data_type *A, data_type *B, size_t N) {
    size_t i = i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        B[i] = A[i];
    }
}

__global__ void RRvectorDivide(const scalar_type *A, const scalar_type *B, scalar_type *C, scalar_type a, size_t N) {
    size_t i = i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        C[i] = a * A[i] / B[i];
    }
}

__global__ void ComplexNorm(const hipfftDoubleComplex *A, scalar_type *B, size_t N) {
    size_t i = i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        B[i] = A[i].x * A[i].x + A[i].y * A[i].y;
    }
}

__global__ void DoubleAbsolute(const scalar_type *A, scalar_type *B, size_t N) {
    size_t i = i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        B[i] = fabs(A[i]);
    }
}

// __global__ void RCvectorMultiply(const scalar_type *A, const hipfftDoubleComplex *X, const hipfftDoubleComplex *Z, scalar_type a, size_t N){
//     size_t i = i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
//
//     if (i < N) {
//         C[i] = a * A[i] * B[i];
//     }
// }

// __global__ void RvectorReciprocal(scalar_type *A, size_t N) {
//     size_t i = i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
//
//     if (i < N) {
//         A[i] = 1. / A[i];
//     }
// }

// same but out of place
__global__ void RvectorReciprocal(const scalar_type *A, scalar_type *B, size_t N) {
    size_t i = i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        B[i] = 1. / A[i];
    }
}



// // equivalent of a*X + Y: I interpret complex vector as double, in-place (X is modified), a is double
// __global__ void axpyDouble( scalar_type *X, const scalar_type *Y, scalar_type a, size_t N) {
//     size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
//
//     if (i < N) {
//         X[i] = a*X[i] + Y[i];
//     }
// }

__global__ void VecInit( scalar_type *X, scalar_type a, size_t N) {
    size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        X[i] = a;
    }
}

__global__ void VecInitComplex( data_type *X, data_type a, size_t N) {
    size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        X[i] = a;
    }
}

// equivalent of a*X + b*Y: I interpret complex vector as double, out-of-place (unless Z = X), a is double
__global__ void axpyDouble( scalar_type *X,  scalar_type *Y, scalar_type *Z, scalar_type a, scalar_type b, size_t N) {
    size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        Z[i] = a*X[i] + b*Y[i];
        // Y[i] = 0.0*Y[i];
    }
}

// equivalent of a*X + b*Y: I use complex vectors, out-of-place (unless Z = X), a is double
// __global__ void axpyComplex( const hipfftDoubleComplex *X, const hipfftDoubleComplex *Y, hipfftDoubleComplex *Z, scalar_type a, scalar_type b, size_t N) {
//     size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
//
//     if (i < N) {
//         Z[i].x = a*X[i].x + b*Y[i].x;
//         Z[i].y = a*X[i].y + b*Y[i].y;
//     }
// }

__global__ void axpyComplex( const data_type *X, data_type *Y, data_type *Z, scalar_type a, scalar_type b, size_t N) {
    size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        Z[i] = a*X[i] + b*Y[i];
        // Y[i] = data_type(0.0,0.0);
    }
}

__global__ void axpy5ComplexAssign( data_type *A, data_type *B, data_type *C, data_type *D, data_type *E, scalar_type a, scalar_type b, scalar_type c, scalar_type d, scalar_type e, size_t N) {
    // real Y = mu_j*Uc(nv,k,j,i) + nu_j*Uc1(nv,k,j,i);
    // Uc1(nv,k,j,i) = Uc(nv,k,j,i);
    // Uc <- Y + (1.0 - mu_j - nu_j)*Uc0 + dt_hyp*mu_tilde_j*dU +  gamma_j*dt_hyp*dU0;
    size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    data_type Y = data_type(0.0,0.0);

    if (i < N) {
        Y = a*A[i] + b*B[i];
        B[i] = A[i];
        A[i] = Y + c*C[i] + d*D[i] + e*E[i];
    }
}

__global__ void addReset( const data_type *X, data_type *Y, data_type *Z, scalar_type a, scalar_type b, size_t N) {
    size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        Z[i] = a*X[i] + b*Y[i];
        Y[i] = data_type(0.0,0.0);
    }
}

// computes a * nabla X, where X is complex vector, out-of-place (unless Z = X), a is double
// __global__ void nablaOp( scalar_type *kx, scalar_type *ky, scalar_type *kz, hipfftDoubleComplex *X, hipfftDoubleComplex *Z, scalar_type a, size_t N, int flag) {
//     size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
//
//     if ( flag == 0 ){ // overwrite i-th element
//         if (i < N) {
//             Z[i].x = - a * (kx[i] * kx[i] + ky[i] * ky[i] + kz[i] * kz[i] ) * X[i].x;
//             Z[i].y = - a * (kx[i] * kx[i] + ky[i] * ky[i] + kz[i] * kz[i] ) * X[i].y;
//         }
//     }
//     else if ( flag == 1) { // accumulate to i-th element
//         if (i < N) {
//             Z[i].x += - a * (kx[i] * kx[i] + ky[i] * ky[i] + kz[i] * kz[i] ) * X[i].x;
//             Z[i].y += - a * (kx[i] * kx[i] + ky[i] * ky[i] + kz[i] * kz[i] ) * X[i].y;
//         }
//     }
//
// }

// __global__ void scalarDissipation( const scalar_type *d_all_kvec, const data_type *X, scalar_type *Z, size_t N) {
//     size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
//     // int vars.KX = 0; int vars.KY = 1; int vars.KZ = 2;
//     // this is the imaginary unit
//     // data_type imI = data_type(0.0,1.0);
//
//
//     if (i < N) {
//         Z[i] = - (d_all_kvec[vars.KX * N + i] * d_all_kvec[vars.KX * N + i] + d_all_kvec[vars.KY * N + i] * d_all_kvec[vars.KY * N + i] + d_all_kvec[vars.KZ * N + i] * d_all_kvec[vars.KZ * N + i] ) * (X[i].real() * X[i].real() + X[i].imag() * X[i].imag());
//     }
//
//
// }
