#include <hipfft/hipfftXt.h>
#include "define_types.hpp"

#include "cufft_routines.hpp"
#include "cuda_kernels_generic.hpp"
#include "spooky.hpp"
#include "supervisor.hpp"
// hipfftHandle plan_r2c{}, plan_c2r{};

hipfftHandle plan_r2c{};
hipfftHandle plan_c2r{};

// extern const int threadsPerBlock;

void r2c_fft(void *r_data_in, void *c_data_out) {

    // Execute the plan_r2c
    CUFFT_CALL(hipfftXtExec(plan_r2c, r_data_in, c_data_out, HIPFFT_FORWARD));


};

void r2c_fft(void *r_data_in, void *c_data_out, Supervisor *supervisor) {

    // increase FFT count
    supervisor->NumFFTs += 1;

    hipEventRecord(supervisor->start);
    // Execute the plan_r2c
    CUFFT_CALL(hipfftXtExec(plan_r2c, r_data_in, c_data_out, HIPFFT_FORWARD));

    hipEventRecord(supervisor->stop);
    hipEventSynchronize(supervisor->stop);
    supervisor->updateFFTtime();


};

void c2r_fft(void *c_data_in, void *r_data_out) {

    // Scale complex results
    int dimGrid, dimBlock;
    dimGrid = (ntotal + threadsPerBlock - 1) / threadsPerBlock;
    dimBlock = threadsPerBlock;

    scaleKernel<<<dimGrid, dimBlock>>>(reinterpret_cast<hipfftDoubleComplex *>(c_data_in), (double) 1./(fft_size[0] * fft_size[1] * fft_size[2]), fft_size[0] * fft_size[1] * ((fft_size[2] / 2) + 1));
    //CUDA_RT_CALL( hipPeekAtLastError() );
    //CUDA_RT_CALL( hipDeviceSynchronize() );

    // Execute the plan_c2r
    CUFFT_CALL(hipfftXtExec(plan_c2r, c_data_in, r_data_out, HIPFFT_BACKWARD));



};

void c2r_fft(void *c_data_in, void *r_data_out, Supervisor *supervisor) {

    // increase FFT count
    supervisor->NumFFTs += 1;
    hipEventRecord(supervisor->start);

    // Scale complex results
    int dimGrid, dimBlock;
    dimGrid = (ntotal + threadsPerBlock - 1) / threadsPerBlock;
    dimBlock = threadsPerBlock;

    scaleKernel<<<dimGrid, dimBlock>>>(reinterpret_cast<hipfftDoubleComplex *>(c_data_in), (double) 1./(fft_size[0] * fft_size[1] * fft_size[2]), fft_size[0] * fft_size[1] * ((fft_size[2] / 2) + 1));
    //CUDA_RT_CALL( hipPeekAtLastError() );
    //CUDA_RT_CALL( hipDeviceSynchronize() );

    // Execute the plan_c2r
    CUFFT_CALL(hipfftXtExec(plan_c2r, c_data_in, r_data_out, HIPFFT_BACKWARD));

    hipEventRecord(supervisor->stop);
    hipEventSynchronize(supervisor->stop);
    supervisor->updateFFTtime();

};

void init_plan(const size_t *fft_size) {
    // Initiate cufft plans, one for r2c and one for c2r

    CUFFT_CALL(hipfftCreate(&plan_r2c));
    CUFFT_CALL(hipfftCreate(&plan_c2r));

    // Create the plans
    size_t workspace_size;
    CUFFT_CALL(hipfftMakePlan3d(plan_r2c, fft_size[0], fft_size[1], fft_size[2], HIPFFT_D2Z, &workspace_size));
    CUFFT_CALL(hipfftMakePlan3d(plan_c2r, fft_size[0], fft_size[1], fft_size[2], HIPFFT_Z2D, &workspace_size));


};

void finish_cufft() {
    CUFFT_CALL(hipfftDestroy(plan_r2c));
    CUFFT_CALL(hipfftDestroy(plan_c2r));
};
