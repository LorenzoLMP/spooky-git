#include <hipfft/hipfftXt.h>
#include "common.hpp"
#include "cufft_routines.hpp"
#include "cuda_kernels_generic.hpp"
#include "supervisor.hpp"

// hipfftHandle plan_r2c{}, plan_c2r{};

hipfftHandle plan_r2c{};
hipfftHandle plan_c2r{};

// extern const int threadsPerBlock;

void r2c_fft(void *r_data_in, void *c_data_out) {

    // Execute the plan_r2c
    CUFFT_CALL(hipfftXtExec(plan_r2c, r_data_in, c_data_out, HIPFFT_FORWARD));


};


void c2r_fft(void *c_data_in, void *r_data_out) {

    // Scale complex results
    int dimGrid, dimBlock;
    dimGrid = (grid.NTOTAL + threadsPerBlock - 1) / threadsPerBlock;
    dimBlock = threadsPerBlock;

    scaleKernel<<<dimGrid, dimBlock>>>(reinterpret_cast<hipfftDoubleComplex *>(c_data_in), (double) 1./(grid.FFT_SIZE[0] * grid.FFT_SIZE[1] * grid.FFT_SIZE[2]), grid.FFT_SIZE[0] * grid.FFT_SIZE[1] * ((grid.FFT_SIZE[2] / 2) + 1));
    //CUDA_RT_CALL( hipPeekAtLastError() );
    //CUDA_RT_CALL( hipDeviceSynchronize() );

    // Execute the plan_c2r
    CUFFT_CALL(hipfftXtExec(plan_c2r, c_data_in, r_data_out, HIPFFT_BACKWARD));



};


void r2c_fft(void *r_data_in, void *c_data_out, Supervisor *supervisor) {

    // increase FFT count
    supervisor->NumFFTs += 1;

    hipEventRecord(supervisor->start);
    // Execute the plan_r2c
    CUFFT_CALL(hipfftXtExec(plan_r2c, r_data_in, c_data_out, HIPFFT_FORWARD));

    hipEventRecord(supervisor->stop);
    hipEventSynchronize(supervisor->stop);
    supervisor->updateFFTtime();


};


void c2r_fft(void *c_data_in, void *r_data_out, Supervisor *supervisor) {

    // increase FFT count
    supervisor->NumFFTs += 1;
    hipEventRecord(supervisor->start);

    // Scale complex results
    int dimGrid, dimBlock;
    dimGrid = (grid.NTOTAL + threadsPerBlock - 1) / threadsPerBlock;
    dimBlock = threadsPerBlock;

    scaleKernel<<<dimGrid, dimBlock>>>(reinterpret_cast<hipfftDoubleComplex *>(c_data_in), (double) 1./(grid.FFT_SIZE[0] * grid.FFT_SIZE[1] * grid.FFT_SIZE[2]), grid.FFT_SIZE[0] * grid.FFT_SIZE[1] * ((grid.FFT_SIZE[2] / 2) + 1));
    //CUDA_RT_CALL( hipPeekAtLastError() );
    //CUDA_RT_CALL( hipDeviceSynchronize() );

    // Execute the plan_c2r
    CUFFT_CALL(hipfftXtExec(plan_c2r, c_data_in, r_data_out, HIPFFT_BACKWARD));

    hipEventRecord(supervisor->stop);
    hipEventSynchronize(supervisor->stop);
    supervisor->updateFFTtime();

};


void init_plan(const size_t *fft_size) {
    // Initiate cufft plans, one for r2c and one for c2r

    CUFFT_CALL(hipfftCreate(&plan_r2c));
    CUFFT_CALL(hipfftCreate(&plan_c2r));

    // Create the plans
    size_t workspace_size;
    CUFFT_CALL(hipfftMakePlan3d(plan_r2c, fft_size[0], fft_size[1], fft_size[2], HIPFFT_D2Z, &workspace_size));
    CUFFT_CALL(hipfftMakePlan3d(plan_c2r, fft_size[0], fft_size[1], fft_size[2], HIPFFT_Z2D, &workspace_size));


};

void finish_cufft() {
    CUFFT_CALL(hipfftDestroy(plan_r2c));
    CUFFT_CALL(hipfftDestroy(plan_c2r));
};


void Complex2RealFields(data_type* ComplexField_in, int num_fields){

    // version with in-place transform
    // compute FFTs from complex to real fields
    for (int n = 0; n < num_fields; n++){
        c2r_fft(ComplexField_in + n * grid.NTOTAL_COMPLEX,  ((scalar_type*) ComplexField_in) + n * 2*grid.NTOTAL_COMPLEX);
    }

}

void Complex2RealFields(data_type* ComplexField_in, scalar_type* RealField_out, int num_fields){

    // assign fields to [num_fields] tmparray (memory block starts at d_all_tmparray)
    int blocksPerGrid = ( num_fields * grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
    ComplexVecAssign<<<blocksPerGrid, threadsPerBlock>>>(ComplexField_in, (data_type*) RealField_out, num_fields * grid.NTOTAL_COMPLEX);

    // compute FFTs from complex to real fields
    for (int n = 0; n < num_fields; n++){
        c2r_fft((data_type*) RealField_out + n * grid.NTOTAL_COMPLEX,  RealField_out + n * 2*grid.NTOTAL_COMPLEX);
    }

}