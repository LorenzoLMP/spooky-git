#include "define_types.hpp"
#include "fields.hpp"
#include "cufft_routines.hpp"
#include "spooky.hpp"
#include "common.hpp"
#include "cuda_kernels.hpp"
#include "cuda_kernels_generic.hpp"
#include "parameters.hpp"

void Fields::AnisotropicConduction() {

    int blocksPerGrid;

#ifdef BOUSSINESQ
#ifdef MHD
#ifdef ANISOTROPIC_DIFFUSION

    // not needed anymore!
    // assign Bx, By, Bz to first 3 scratch arrays
    // blocksPerGrid = ( 3 * ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    // ComplexVecAssign<<<blocksPerGrid, threadsPerBlock>>>((hipfftDoubleComplex *)d_all_fields + ntotal_complex * BX, (hipfftDoubleComplex *)d_all_tmparray, 3 * ntotal_complex);


    // Bx, By, Bz real fields are already in the 4-5-6 tmp arrays
    // compute gradient of theta and assign it to next 3 scratch arrays [num_fields -- num_fields + 3] (the first num_fields arrays are reserved for the real-valued fields)
    blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    Gradient<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)wavevector.d_all_kvec, (data_type *) d_farray[TH], (data_type *)d_all_tmparray + num_fields * ntotal_complex, ntotal_complex);
    // compute complex to real iFFTs
    for (int n = num_fields; n < num_fields + 3; n++){
        c2r_fft(d_tmparray[n], d_tmparray_r[n]);
    }
    // compute the scalar B grad theta (real space) and assign it to 7th scratch array
    blocksPerGrid = ( 2 * ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    ComputeBGradTheta<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *) d_tmparray_r[BX], (scalar_type *) d_all_tmparray + 2 * ntotal_complex * num_fields, (scalar_type *) d_tmparray_r[num_fields + 3], 2 * ntotal_complex);
    // compute the anisotropic heat flux and put it in the 3-4-5 tmp arrays
    ComputeAnisotropicHeatFlux<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *) d_tmparray_r[BX], (scalar_type *) d_tmparray_r[num_fields + 3], (scalar_type *) d_tmparray_r[num_fields], param->OmegaT2, (1./param->reynolds_ani), 2 * ntotal_complex, STRAT_DIR);
    // take fourier transforms of the heat flux
    for (int n = num_fields ; n < num_fields + 3; n++) {
        r2c_fft(d_tmparray_r[n], d_tmparray[n]);
    }
    // take divergence of heat flux
    blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    DivergenceMask<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)wavevector.d_all_kvec, (data_type *) d_tmparray[num_fields], (data_type *) d_all_dfields + TH * ntotal_complex, (scalar_type *)wavevector.d_mask, ntotal_complex, ADD);

#endif // ANISOTROPIC_DIFFUSION
#endif // MHD
#endif // BOUSSINESQ

}
