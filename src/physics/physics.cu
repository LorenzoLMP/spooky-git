#include "define_types.hpp"
#include "physics.hpp"
// #include "timestepping.hpp"
// #include "cufft_routines.hpp"
#include "spooky.hpp"
#include "common.hpp"
#include "cublas_routines.hpp"
#include "cuda_kernels.hpp"
#include "parameters.hpp"
// #include "inputoutput.hpp"
#include "fields.hpp"
#include <hip/hip_runtime.h>
// #include <hipfft/hipfftXt.h>
// #include "spooky.hpp"
#include "cufft_utils.h"
// #include "define_types.hpp"


Physics::Physics(Supervisor &sup){

    supervisor = &sup;

}

Physics::~Physics(){

}
