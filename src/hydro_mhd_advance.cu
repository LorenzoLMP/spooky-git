#include "define_types.hpp"
#include "fields.hpp"
#include "cufft_routines.hpp"
#include "spooky.hpp"
#include "hydro_mhd_advance.hpp"
#include "cublas_routines.hpp"
// #include "compute_timestep.hpp"
#include "cuda_kernels.hpp"
#include "cuda_kernels_generic.hpp"


const double gammaRK[3] = {8.0 / 15.0 , 5.0 / 12.0 , 3.0 / 4.0};
const double xiRK[2] = {-17.0 / 60.0 , -5.0 / 12.0};
hipblasStatus_t stat;
// extern int threadsPerBlock;


void Fields::RungeKutta3( double t, double t_end, Parameters  *param) {
    NVTX3_FUNC_RANGE();
    double dt_RK = 0.0;
    int blocksPerGrid = (2 * ntotal_complex * num_fields + threadsPerBlock - 1) / threadsPerBlock;
    int RK_step = 0;
    current_step += 1;

    // compute_dt( );

    // std::printf("...Computing dfield\n");
    compute_dfield(RK_step++, param);
    if ( t + current_dt > t_end) current_dt = t_end - t;
    dt_RK = current_dt; // in theory one can do strang splitting so dt_RK can be 1/2 dt
#ifdef DEBUG
    std::printf("RK, 1st step:\n");
    std::printf("After compute dfield, RK, 1st step:\n");
    // print_device_values();
    if (current_step == 1 || current_step % 100 == 0 ) std::printf("t: %.5e \t dt: %.5e \n",t,dt_RK);
    if (current_step == 1 || current_step % 100 == 0 ) print_device_values();
#endif



    // snooopy code
    // for( n = 0 ; n < fld.nfield ; n++) {
    //     for( i = 0 ; i < NTOTAL_COMPLEX ; i++) {
    //         fld.farray[n][i] = fld.farray[n][i] + gammaRK[0] * dfld.farray[n][i] * dt;
    //         fld1.farray[n][i] = fld.farray[n][i] + xiRK[0] * dfld.farray[n][i] * dt;
    //     }
    // }

    // d_all_fields = d_all_fields + gammaRK[0] * dt * d_all_dfields;
    axpyDouble<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *)d_all_fields, (scalar_type *)d_all_dfields, (scalar_type *)d_all_fields, (scalar_type) 1.0, gammaRK[0]*dt_RK,  2 * ntotal_complex * num_fields);
    // // d_all_scrtimestep = d_all_fields + xiRK[0] * dt * d_all_dfields;
    axpyDouble<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *)d_all_fields, (scalar_type *)d_all_dfields, (scalar_type *)d_all_scrtimestep, (scalar_type) 1.0, xiRK[0]*dt_RK,  2 * ntotal_complex * num_fields);

#ifdef DEBUG
    std::printf("After 1st RK:\n");
    // print_device_values();
    std::printf("RK, 2nd step:\n");
#endif
    // std::printf("...Computing dfield\n");
    compute_dfield(RK_step++, param);
    // for( i = 0 ; i < NTOTAL_COMPLEX ; i++) {
    //         fld.farray[n][i] = fld1.farray[n][i] + gammaRK[1] * dfld.farray[n][i] * dt;
    //         fld1.farray[n][i] = fld.farray[n][i] + xiRK[1] * dfld.farray[n][i] * dt;
    //     }

    // d_all_fields = d_all_scrtimestep + gammaRK[1] * dt * d_all_dfields;
    axpyDouble<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *)d_all_scrtimestep, (scalar_type *)d_all_dfields, (scalar_type *)d_all_fields, (scalar_type) 1.0, gammaRK[1]*dt_RK,  2 * ntotal_complex * num_fields);
    // d_all_scrtimestep = d_all_fields + xiRK[1] * dt * d_all_dfields;
    axpyDouble<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *)d_all_fields, (scalar_type *)d_all_dfields, (scalar_type *)d_all_scrtimestep, (scalar_type) 1.0, xiRK[1]*dt_RK,  2 * ntotal_complex * num_fields);

#ifdef DEBUG
    std::printf("After 2nd RK:\n");
    // print_device_values();
    std::printf("RK, 3rd step:\n");
#endif
    // std::printf("...Computing dfield\n");
    compute_dfield(RK_step++, param);
    // for( i = 0 ; i < NTOTAL_COMPLEX ; i++) {
    //     fld.farray[n][i] = fld1.farray[n][i] + gammaRK[2] * dfld.farray[n][i] * dt;
    // }
    // d_all_fields = d_all_scrtimestep + gammaRK[2] * dt * d_all_dfields;
    axpyDouble<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *)d_all_scrtimestep, (scalar_type *)d_all_dfields, (scalar_type *)d_all_fields, (scalar_type) 1.0, gammaRK[2]*dt_RK,  2 * ntotal_complex * num_fields);


    return ;


}




// double Fields::advance_timestep( double t, double t_end, int* p_step) {
//     NVTX3_FUNC_RANGE();
//     double dt = 0.0;
//     // int step = 0;
//
//     // std::printf("Computing dt\n");
//     compute_dt( &dt );
//
//     if ( t + dt > t_end) dt = t_end - t;
//     // std::printf("dt: %.2e \n",dt);
//     // one can implement different solvers...
//     // RungeKutta3((double)0.5);
//     RungeKutta3((double)dt);
//     // std::printf("Updating n. step\n");
//     *p_step = *p_step + 1;
//     return dt;
// }
