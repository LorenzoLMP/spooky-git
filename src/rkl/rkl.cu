#include "hip/hip_runtime.h"
#include "common.hpp"
#include "rkl.hpp"
// #include "cufft_routines.hpp"
#include "cublas_routines.hpp"
#include "cuda_kernels.hpp"
#include "cuda_kernels_generic.hpp"
#include "parameters.hpp"
// #include "inputoutput.hpp"
#include "fields.hpp"
#include "physics.hpp"
#include <hip/hip_runtime.h>
#include "cufft_utils.h"

#include "supervisor.hpp"
#include "timestepping.hpp"
// #include <cstdlib>
#include <math.h>


RKLegendre::RKLegendre(Parameters &p_in, Supervisor &sup_in) {
    // param = &p_in;
    // fields = &f_in;

    supervisor_ptr = &sup_in;

    dt = 0.0;
    stage = 0;
    cfl_rkl = p_in.cfl_par;
    rmax_par = p_in.safety_sts;

    sts_algorithm = p_in.sts_algorithm;


    std::vector<std::string> sts_variables = p_in.sts_variables;

    num_sts_vars = sts_variables.size();
    // sts_variables_index contains the indices
    // of the variables that should be evolved
    // with supertimestepping
    sts_variables_index.resize(num_sts_vars);
    // sts_variables_pos says whether that given variable
    // should be ststepped (in which case is holds
    // the value of the index in the sts_variables list)
    // or not (in which case it is -1)
    sts_variables_pos.resize(vars.NUM_FIELDS);
    for (int j = 0; j < vars.NUM_FIELDS; j++) {
        // the variable is not in the sts list
        sts_variables_pos[j] = -1;
        for (int i = 0; i < num_sts_vars; i++) {
            if (!sts_variables[i].compare(vars.VAR_LIST[j])) {
                // the variable is in the sts list
                sts_variables_index[i] = j;
                sts_variables_pos[j] = i;
                break ;
            }
        }
    }
    std::printf("sts variables pos \n");
    for (int j = 0; j < vars.NUM_FIELDS; j++) {
        std::cout << sts_variables_pos[j] << std::endl;
    }

    std::printf("sts variables index \n");
    for (int j = 0; j < num_sts_vars; j++) {
        std::cout << sts_variables_index[j] << std::endl;
    }

    blocksPerGrid = ( num_sts_vars * grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
    // this is the mega array that contains intermediate fields during multi-stage timestepping
    // std::printf("num fields ts: %d \n", vars.NUM_FIELDS);
    if (!sts_algorithm.compare(std::string("sts"))) {

        // sts_algorithm = "sts";
        ts = new double[STS_MAX_STEPS];
        for (int i = 0; i < STS_MAX_STEPS; i++){
            ts[i] = 0.0;
        }

        std::printf("num rkl scratch arrays: %d \n",num_sts_vars);
        CUDA_RT_CALL(hipMalloc(&d_all_dU, (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX * num_sts_vars));

        VecInitComplex<<<blocksPerGrid, threadsPerBlock>>>(d_all_dU,  data_type(0.0,0.0), grid.NTOTAL_COMPLEX * num_sts_vars);

        d_farray_dU  = new data_type*[num_sts_vars];

        for (int i = 0; i < num_sts_vars; i++){
            d_farray_dU[i]   = d_all_dU + i*grid.NTOTAL_COMPLEX;
        }

    }
    else if (!sts_algorithm.compare(std::string("rkl3"))) {
        // sts_algorithm = "rkl3";
        std::printf("num rkl scratch arrays: %d \n",4*num_sts_vars);

        CUDA_RT_CALL(hipMalloc(&d_all_dU, (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX * num_sts_vars));
        CUDA_RT_CALL(hipMalloc(&d_all_dU0, (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX * num_sts_vars));
        CUDA_RT_CALL(hipMalloc(&d_all_Uc0, (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX * num_sts_vars));
        CUDA_RT_CALL(hipMalloc(&d_all_Uc1, (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX * num_sts_vars));

        VecInitComplex<<<blocksPerGrid, threadsPerBlock>>>(d_all_dU,  data_type(0.0,0.0), grid.NTOTAL_COMPLEX * num_sts_vars);
        VecInitComplex<<<blocksPerGrid, threadsPerBlock>>>(d_all_dU0, data_type(0.0,0.0), grid.NTOTAL_COMPLEX * num_sts_vars);
        VecInitComplex<<<blocksPerGrid, threadsPerBlock>>>(d_all_Uc0, data_type(0.0,0.0), grid.NTOTAL_COMPLEX * num_sts_vars);
        VecInitComplex<<<blocksPerGrid, threadsPerBlock>>>(d_all_Uc1, data_type(0.0,0.0), grid.NTOTAL_COMPLEX * num_sts_vars);

        d_farray_dU  = new data_type*[num_sts_vars];
        d_farray_dU0 = new data_type*[num_sts_vars];
        d_farray_Uc0 = new data_type*[num_sts_vars];
        d_farray_Uc1 = new data_type*[num_sts_vars];

        for (int i = 0; i < num_sts_vars; i++){
            d_farray_dU[i]   = d_all_dU + i*grid.NTOTAL_COMPLEX;
            d_farray_dU0[i]   = d_all_dU0 + i*grid.NTOTAL_COMPLEX;
            d_farray_Uc0[i]   = d_all_Uc0 + i*grid.NTOTAL_COMPLEX;
            d_farray_Uc1[i]   = d_all_Uc1 + i*grid.NTOTAL_COMPLEX;
        }
    }

    // std::printf("blocksPerGrid: %d \n",blocksPerGrid);

}

RKLegendre::~RKLegendre(){
    if (!sts_algorithm.compare(std::string("sts"))) {
        CUDA_RT_CALL(hipFree(d_all_dU));
        delete d_farray_dU;
        delete ts;
    }
    else if (!sts_algorithm.compare(std::string("rkl3"))) {

        CUDA_RT_CALL(hipFree(d_all_dU));
        CUDA_RT_CALL(hipFree(d_all_dU0));
        CUDA_RT_CALL(hipFree(d_all_Uc0));
        CUDA_RT_CALL(hipFree(d_all_Uc1));
        delete d_farray_dU, d_farray_dU0, d_farray_Uc0, d_farray_Uc1;
    }

}

void RKLegendre::compute_cycle(data_type* complex_Fields, scalar_type* real_Buffer){

    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;

    if (!param_ptr->sts_algorithm.compare(std::string("sts"))) {
        compute_cycle_STS(complex_Fields, real_Buffer);
    }
    else if (!param_ptr->sts_algorithm.compare(std::string("rkl3"))) {
        compute_cycle_RKL(complex_Fields, real_Buffer);
    }

}


double STS_CorrectTimeStep(int n0, double dta, double STS_NU);
double STS_FindRoot(double dt_exp, double dT, double STS_NU);
void STS_ComputeSubSteps(double dtex, double* tau, int N, double STS_NU);


void RKLegendre::compute_cycle_STS(data_type* complex_Fields, scalar_type* real_Buffer){

    std::shared_ptr<Fields> fields_ptr = supervisor_ptr->fields_ptr;
    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;
    std::shared_ptr<TimeStepping> timestep_ptr = supervisor_ptr->timestep_ptr;
    std::shared_ptr<Physics> phys_ptr = supervisor_ptr->phys_ptr;

    double dt_hyp = timestep_ptr->current_dt;
    double dt_par = timestep_ptr->dt_par;
    double dt_par_corr = dt_par;

    // std::printf("now in supertimestepping function");
    // int i;
    int nv, n, m;
    double N;
    // int nv_indx, nvar_rkl;
    double tau;


    tau = dt_par;

    m = 0;
    n = STS_MAX_STEPS;
    while (m < n){

        N = STS_FindRoot(dt_par, dt_hyp, STS_NU);
        N = floor(N+1.0);
        n = (int)N;

        if (param_ptr->debug > 0) {
            std::printf("STS::::: number of STS subcycles: %d \n",n);
        }

        if (n > 1){
            dt_par_corr = STS_CorrectTimeStep(n, dt_hyp, STS_NU);
            if (param_ptr->debug > 1) {
                std::printf("STS::::: dt_par_corr: %4.e \n",dt_par_corr);
            }
            STS_ComputeSubSteps(dt_par_corr, ts, n, STS_NU);
        }
        if (n == 1) ts[0] = dt_hyp;
        tau = ts[n-m-1];
        if (param_ptr->debug > 0) {
            std::printf("STS::::: tau: %4.e \n",tau);
        }


        // anisotropic_conduction( rhs, fldi);
        // phys_ptr->AnisotropicConduction(fields, param, (data_type *) fields_ptr->d_farray[vars.TH], (data_type *) d_farray_dU[vars.TH]);
        // this is for all parabolic terms
        phys_ptr->ParabolicTermsSTS(complex_Fields, real_Buffer, d_all_dU);


        // this is the index corresponding
        // to the position in the VAR array of a given
        // sts_var
        int var_idx;

        blocksPerGrid = (grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
        for (nv = 0; nv < num_sts_vars; nv++){
            var_idx = sts_variables_index[nv];
            addReset<<<blocksPerGrid, threadsPerBlock>>>( complex_Fields + var_idx * grid.NTOTAL_COMPLEX,  d_farray_dU[nv],  complex_Fields + var_idx * grid.NTOTAL_COMPLEX, 1.0, tau, grid.NTOTAL_COMPLEX);
        }
        CUDA_RT_CALL( hipDeviceSynchronize() );

        m++;
    }

}


void RKLegendre::compute_cycle_RKL(data_type* complex_Fields, scalar_type* real_Buffer){

    std::shared_ptr<Fields> fields_ptr = supervisor_ptr->fields_ptr;
    std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;
    std::shared_ptr<TimeStepping> timestep_ptr = supervisor_ptr->timestep_ptr;
    std::shared_ptr<Physics> phys_ptr = supervisor_ptr->phys_ptr;

    double dt_hyp = timestep_ptr->current_dt;
    double dt_par = timestep_ptr->dt_par;
    double time = timestep_ptr->current_time;

    // std::printf("now in supertimestepping function");


    // tau is dt_hyp
    // static Data_Arr Y_jm1, Y_jm2, MY_jm1, MY_0;
    // in idefix they correspond to:
    // Y_jm1  --> Uc0    // field step j-1
    // Y_jm2  --> Uc1   // field step j-2
    // MY_jm1 --> dU    // dfield step j-1
    // MY_0   --> dU0   // dfield step 0
    // static double **v;
    double s_str;                          /* The "s" parameter */

    // int i;
    int nv, s, s_RKL = 0;
    double scrh;
    // int nv_indx, var_list[vars.NUM_FIELDS], nvar_rkl;
    double mu_j, nu_j, mu_tilde_j, gamma_j;
    // data_type Y;
    double a_jm1, b_j, b_jm1, b_jm2, w1;


    scrh  = dt_hyp/dt_par;                      /*  Solution of quadratic Eq.   */
    s_str =   4.0*(1.0 + 2.0*scrh)           /*  4*tau/dt_exp = s^2 + s - 2  */
            /(1.0 + sqrt(9.0 + 16.0*scrh));

    s_RKL = 1 + int(s_str);
    if (param_ptr->debug > 0) {
        std::printf("RKL::::: number of RKL subcycles: %d \n",s_RKL);
    }
    w1 = 4.0/(s_RKL*s_RKL + s_RKL - 2.0);
    mu_tilde_j = w1/3.0;

    b_j = b_jm1 = b_jm2 = 1.0/3.0;
    a_jm1 = 1.0 - b_jm1;


    // initialize temp fields
    // MY_0 <- parabolicRHS(d_farray[vars.TH])
    blocksPerGrid = ( num_sts_vars * grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
    VecInitComplex<<<blocksPerGrid, threadsPerBlock>>>((data_type *)d_all_dU0, data_type(0.0,0.0), num_sts_vars * grid.NTOTAL_COMPLEX);

    // this is only for temperature
    // phys_ptr->AnisotropicConduction(fields, param, (data_type *) fields_ptr->d_farray[vars.TH], (data_type *) d_farray_dU0[vars.TH]);

    // this is for all parabolic terms
    phys_ptr->ParabolicTermsSTS(complex_Fields, real_Buffer, d_all_dU0);
    // phys_ptr->ParabolicTerms(complex_Fields, real_Buffer, d_all_dU0);

    // this is the index corresponding
    // to the position in the VAR array of a given
    // sts_var
    int var_idx;

    for (nv = 0; nv < num_sts_vars; nv++){
        var_idx = sts_variables_index[nv];

        // Y_jm1 <- d_farray[vars.TH]
        blocksPerGrid = ( grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;

        ComplexVecAssign<<<blocksPerGrid, threadsPerBlock>>>(complex_Fields + var_idx * grid.NTOTAL_COMPLEX, d_farray_Uc0[nv], grid.NTOTAL_COMPLEX);

        // Y_jm2 <- d_farray[vars.TH]
        ComplexVecAssign<<<blocksPerGrid, threadsPerBlock>>>(complex_Fields + var_idx * grid.NTOTAL_COMPLEX, d_farray_Uc1[nv], grid.NTOTAL_COMPLEX);

        // Y_jm1 (d_farray[vars.TH]) <- Y_jm2 + mu_tilde_j*dt_hyp*MY_0
        axpyComplex<<<blocksPerGrid, threadsPerBlock>>>( d_farray_Uc1[nv],  d_farray_dU[nv],  complex_Fields + var_idx * grid.NTOTAL_COMPLEX, 1.0, mu_tilde_j*dt_hyp,  grid.NTOTAL_COMPLEX);
    }

    /* s loop */
    s = 1;
    // g_time = t0 + 0.25*tau*(s*s+s-2)*w1;
    for (s = 2; s <= s_RKL; s++) {

        mu_j       = (2.*s -1.)/s * b_j/b_jm1;   /* Eq. [17] */
        mu_tilde_j = w1*mu_j;
        gamma_j    = -a_jm1*mu_tilde_j;
        nu_j       = -(s -1.)*b_j/(s*b_jm2);

        b_jm2 = b_jm1;    /* Eq. [16] */
        b_jm1 = b_j;
        a_jm1 = 1.0 - b_jm1;
        b_j   = 0.5*(s*s+3.0*s)/(s*s+3.0*s+2);

        blocksPerGrid = ( num_sts_vars * grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
        VecInitComplex<<<blocksPerGrid, threadsPerBlock>>>((data_type *)d_all_dU, data_type(0.0,0.0), num_sts_vars * grid.NTOTAL_COMPLEX);

        // phys_ptr->AnisotropicConduction(fields, param, (data_type *) fields_ptr->d_farray[vars.TH], (data_type *) d_farray_dU[vars.TH]);

        phys_ptr->ParabolicTermsSTS(complex_Fields, real_Buffer, d_all_dU);
        // phys_ptr->ParabolicTerms(complex_Fields, real_Buffer, d_all_dU);

        for (nv = 0; nv < num_sts_vars; nv++){
            var_idx = sts_variables_index[nv];

            // MY_j-1 <- parabolicRHS(d_farray[vars.TH])

            blocksPerGrid = ( grid.NTOTAL_COMPLEX + threadsPerBlock - 1) / threadsPerBlock;
            // real Y = mu_j*Uc(nv,k,j,i) + nu_j*Uc1(nv,k,j,i);
            // Uc1(nv,k,j,i) = Uc(nv,k,j,i);
            // Uc <- Y + (1.0 - mu_j - nu_j)*Uc0 + dt_hyp*mu_tilde_j*dU +  gamma_j*dt_hyp*dU0;
            axpy5ComplexAssign<<<blocksPerGrid, threadsPerBlock>>>((data_type *) complex_Fields + var_idx * grid.NTOTAL_COMPLEX, (data_type *) d_farray_Uc1[nv], (data_type *) d_farray_Uc0[nv], (data_type *) d_farray_dU[nv], (data_type *) d_farray_dU0[nv], mu_j, nu_j, (1.0 - mu_j - nu_j), dt_hyp*mu_tilde_j,  gamma_j*dt_hyp, grid.NTOTAL_COMPLEX);

            // increment time
            time = timestep_ptr->current_time + 0.25*dt_hyp*(s*s+s-2)*w1;
        }
    }


}


void STS_ComputeSubSteps(double dtex, double* tau, int N, double STS_NU)
/*!
 * Compute the single sub-step sequence (Eq. [2.9]).
 * N must be an integer by now.
 *
 *********************************************************************** */
{
  int i;
  double S=0.0;

  for (i = 0; i < N; i++) {
    tau[i] = dtex / ((-1.0 + STS_NU)*cos(((2.0*i+1.0)*M_PI)/(2.0*N))
                     + 1.0 + STS_NU);
    S += tau[i];
  }
}

/* ********************************************************************* */
double STS_FindRoot(double dt_exp, double dT, double STS_NU)
/*!
 * Find the number of sub-steps N by solving Eq. (2.10) of AAG using a
 * Newton-Raphson scheme.
 * Input to the function are:
 *
 * \param [in]  dt_exp   the explicit time step
 * \param [in]  dt       the super-step.
 *
 *********************************************************************** */
{
  int k;  /* Iteration number */
  double a,b,c, scrh;
  double fN, N, dN, dfN;
  double db_dN, sqrt_nu = sqrt(STS_NU);

  k = 0;
  N = 1.0;
  a = (1.0 - sqrt_nu)/(1.0 + sqrt_nu);
  while(k < 128){
    b     = pow(a,2.0*N);
    c     = (1.0-b)/(1.0+b);    /* round bracket in Eq. [10] in AAG */
    db_dN = 2.0*log(a)*b;
    scrh  = c - N*2.0/((1.0+b)*(1.0+b))*db_dN;

    fN  = dT - 0.5*dt_exp/sqrt_nu*N*c;
    dfN =    - 0.5*dt_exp/sqrt_nu*scrh;
    dN  = fN/dfN;

    N -= dN;
    k++;

    if (fabs(dN) < 1.e-5) return N;
  }
  return -1.0;
}

/* ********************************************************************* */
double STS_CorrectTimeStep(int n0, double dta, double STS_NU)
/*
 *
 *********************************************************************** */
{
  double a,b,c;
  double dtr;

  a = (1.0-sqrt(STS_NU))/(1.0+sqrt(STS_NU));
  b = pow(a,2.0*n0);
  c = (1.0-b)/(1.0+b);

  dtr = dta*2.0*sqrt(STS_NU)/(n0*c);
  return(dtr);
}
