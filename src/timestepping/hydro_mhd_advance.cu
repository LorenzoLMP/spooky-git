#include "define_types.hpp"
#include "timestepping.hpp"
#include "cufft_routines.hpp"
#include "spooky.hpp"
#include "hydro_mhd_advance.hpp"
#include "cublas_routines.hpp"
// #include "compute_timestep.hpp"
#include "cuda_kernels.hpp"
#include "cuda_kernels_generic.hpp"
#include "parameters.hpp"
#include "fields.hpp"


const double gammaRK[3] = {8.0 / 15.0 , 5.0 / 12.0 , 3.0 / 4.0};
const double xiRK[2] = {-17.0 / 60.0 , -5.0 / 12.0};
hipblasStatus_t stat;
// extern int threadsPerBlock;


void TimeStepping::RungeKutta3(Fields &fields, Parameters &param) {
    NVTX3_FUNC_RANGE();

#ifdef DDEBUG
    std::printf("Now entering RungeKutta3 function \n");
#endif

    double dt_RK = 0.0;
    int blocksPerGrid = (2 * ntotal_complex * fields.num_fields + threadsPerBlock - 1) / threadsPerBlock;
    stage_step = 0;
    current_step += 1;

#ifdef DDEBUG
    // std::printf("After 1st RK:\n");
    // print_device_values();
    // std::printf("num_fields : %d \n",fields.num_fields);
    std::printf("RK3, doing step n. %d ...\n",stage_step+1);
#endif

    // compute_dt( );
    // note that the following compute_dfield also compute the new current_dt!!
    compute_dfield(fields, param);
    stage_step++;

    // std::printf("...Computing dfield\n");
    
    if ( current_time + current_dt > param.t_final) current_dt = param.t_final - current_time;
    dt_RK = current_dt; // in theory one can do strang splitting so dt_RK can be 1/2 dt
    
#ifdef DDEBUG
    // std::printf("RK, finished 1st step.\n");
    // std::printf("After compute dfield, RK, 1st step:\n");
    // print_device_values();
    if (current_step == 1 || current_step % 100 == 0 ) std::printf("t: %.5e \t dt: %.5e \n",current_time,dt_RK);
    if (current_step == 1 || current_step % 100 == 0 ) fields.print_device_values();
#endif
    current_time += current_dt;



    // snooopy code
    // for( n = 0 ; n < fld.nfield ; n++) {
    //     for( i = 0 ; i < NTOTAL_COMPLEX ; i++) {
    //         fld.farray[n][i] = fld.farray[n][i] + gammaRK[0] * dfld.farray[n][i] * dt;
    //         fld1.farray[n][i] = fld.farray[n][i] + xiRK[0] * dfld.farray[n][i] * dt;
    //     }
    // }

    // d_all_fields = d_all_fields + gammaRK[0] * dt * d_all_dfields;
    axpyDouble<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *)fields.d_all_fields, (scalar_type *)fields.d_all_dfields, (scalar_type *)fields.d_all_fields, (scalar_type) 1.0, gammaRK[0]*dt_RK,  2 * ntotal_complex * fields.num_fields);
    // // d_all_scrtimestep = d_all_fields + xiRK[0] * dt * d_all_dfields;
    axpyDouble<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *)fields.d_all_fields, (scalar_type *)fields.d_all_dfields, (scalar_type *)d_all_scrtimestep, (scalar_type) 1.0, xiRK[0]*dt_RK,  2 * ntotal_complex * fields.num_fields);

#ifdef DDEBUG
    std::printf("RK3, doing step n. %d ...\n",stage_step+1);
#endif
    // std::printf("...Computing dfield\n");
    compute_dfield(fields, param);
    stage_step++;
    // for( i = 0 ; i < NTOTAL_COMPLEX ; i++) {
    //         fld.farray[n][i] = fld1.farray[n][i] + gammaRK[1] * dfld.farray[n][i] * dt;
    //         fld1.farray[n][i] = fld.farray[n][i] + xiRK[1] * dfld.farray[n][i] * dt;
    //     }

    // d_all_fields = d_all_scrtimestep + gammaRK[1] * dt * d_all_dfields;
    axpyDouble<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *)d_all_scrtimestep, (scalar_type *)fields.d_all_dfields, (scalar_type *)fields.d_all_fields, (scalar_type) 1.0, gammaRK[1]*dt_RK,  2 * ntotal_complex * fields.num_fields);
    // d_all_scrtimestep = d_all_fields + xiRK[1] * dt * d_all_dfields;
    axpyDouble<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *)fields.d_all_fields, (scalar_type *)fields.d_all_dfields, (scalar_type *)d_all_scrtimestep, (scalar_type) 1.0, xiRK[1]*dt_RK,  2 * ntotal_complex * fields.num_fields);

#ifdef DDEBUG
    std::printf("RK3, doing step n. %d ...\n",stage_step+1);
#endif
    // std::printf("...Computing dfield\n");
    compute_dfield(fields, param);
    stage_step++;
    // for( i = 0 ; i < NTOTAL_COMPLEX ; i++) {
    //     fld.farray[n][i] = fld1.farray[n][i] + gammaRK[2] * dfld.farray[n][i] * dt;
    // }
    // d_all_fields = d_all_scrtimestep + gammaRK[2] * dt * d_all_dfields;
    axpyDouble<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *)d_all_scrtimestep, (scalar_type *)fields.d_all_dfields, (scalar_type *)fields.d_all_fields, (scalar_type) 1.0, gammaRK[2]*dt_RK,  2 * ntotal_complex * fields.num_fields);

#ifdef DDEBUG
    std::printf("End of RK3 integrator, t: %.5e \t dt: %.5e \n",current_time,current_dt);
#endif

    return ;


}

