#include "hip/hip_runtime.h"
#include "define_types.hpp"
#include "timestepping.hpp"
#include "fields.hpp"
#include "cufft_routines.hpp"
#include "spooky.hpp"
#include "common.hpp"
#include "cuda_kernels.hpp"
#include "cuda_kernels_generic.hpp"
#include "cublas_routines.hpp"
#include "parameters.hpp"
#include "physics.hpp"

// we are assuming that the fields have been already fft to real and saved in d_tmparray_r
void TimeStepping::compute_dt(Fields &fields, Parameters &param, Physics &phys) {

    NVTX3_FUNC_RANGE();
    double dt;

#ifdef DDEBUG
    std::printf("Now entering compute_dt function \n");
#endif

#ifdef INCOMPRESSIBLE
    double gamma_v = 0.0, gamma_tot = 0.0;
    double maxfx, maxfy, maxfz;

    maxfx=0.0;
    maxfy=0.0;
    maxfz=0.0;

    int idx_max_vx, idx_max_vy, idx_max_vz;
    hipblasStatus_t stat;


    stat = hipblasIdamax(handle0, 2 * ntotal_complex, fields.d_tmparray_r[VX], 1, &idx_max_vx);
    if (stat != HIPBLAS_STATUS_SUCCESS) std::printf("vx max failed\n");
    stat = hipblasIdamax(handle0, 2 * ntotal_complex, fields.d_tmparray_r[VY], 1, &idx_max_vy);
    if (stat != HIPBLAS_STATUS_SUCCESS) std::printf("vy max failed\n");
    stat = hipblasIdamax(handle0, 2 * ntotal_complex, fields.d_tmparray_r[VZ], 1, &idx_max_vz);
    if (stat != HIPBLAS_STATUS_SUCCESS) std::printf("vz max failed\n");


    // index is in fortran convention
    CUDA_RT_CALL(hipMemcpy(&maxfx, &fields.d_tmparray_r[VX][idx_max_vx-1], sizeof(scalar_type), hipMemcpyDeviceToHost));
    CUDA_RT_CALL(hipMemcpy(&maxfy, &fields.d_tmparray_r[VY][idx_max_vy-1], sizeof(scalar_type), hipMemcpyDeviceToHost));
    CUDA_RT_CALL(hipMemcpy(&maxfz, &fields.d_tmparray_r[VZ][idx_max_vz-1], sizeof(scalar_type), hipMemcpyDeviceToHost));


    maxfx=fabs(maxfx);
    maxfy=fabs(maxfy);
    maxfz=fabs(maxfz);



    gamma_v = ( fields.wavevector.kxmax ) * maxfx + fields.wavevector.kymax * maxfy + fields.wavevector.kzmax * maxfz;


#ifdef WITH_ROTATION
    gamma_v += fabs(param.omega) / param.safety_source;
#endif

#ifdef WITH_SHEAR
    gamma_v += fabs(param.shear) / param.safety_source;
#endif

// #ifdef INCOMPRESSIBLE
#ifdef WITH_EXPLICIT_DISSIPATION
	gamma_v += ((fields.wavevector.kxmax )*( fields.wavevector.kxmax )+fields.wavevector.kymax*fields.wavevector.kymax+fields.wavevector.kzmax*fields.wavevector.kzmax) * param.nu;	// CFL condition on viscosity in incompressible regime
#endif
// #endif

#ifdef BOUSSINESQ
    gamma_v += pow(fabs(param.N2), 0.5) / param.safety_source;
#ifdef WITH_EXPLICIT_DISSIPATION
#ifndef ANISOTROPIC_DIFFUSION
    gamma_v += ((fields.wavevector.kxmax )*( fields.wavevector.kxmax )+fields.wavevector.kymax*fields.wavevector.kymax+fields.wavevector.kzmax*fields.wavevector.kzmax) * param.nu_th; // NB: this is very conservative. It should be combined with the condition on nu
#else
    gamma_v += ((fields.wavevector.kxmax )*( fields.wavevector.kxmax )+fields.wavevector.kymax*fields.wavevector.kymax+fields.wavevector.kzmax*fields.wavevector.kzmax) * (1./param.reynolds_ani); //
#endif // ANISOTROPIC_DIFFUSION
#endif // WITH_EXPLICIT_DISSIPATION
#endif // BOUSSINESQ

#ifdef DDEBUG
    if (current_step == 1 || current_step % 100 == 0 ) std::printf("maxfx: %.4e \t maxfy: %.4e \t maxfz: %.4e \t gamma_v: %.4e \n",maxfx,maxfy,maxfz,gamma_v);
#endif

#ifdef MHD
    double gamma_b;
    double maxbx, maxby, maxbz;


    maxbx=0.0;
    maxby=0.0;
    maxbz=0.0;

    int idx_max_bx, idx_max_by, idx_max_bz;
    // hipblasStatus_t stat;

    // here we need to do a c2r transform first when in production

    // c2r_fft(d_farray[BX], d_farray_r[BX]);
    // c2r_fft(d_farray[BY], d_farray_r[BY]);
    // c2r_fft(d_farray[BZ], d_farray_r[BZ]);

    stat = hipblasIdamax(handle0, 2 * ntotal_complex, fields.d_tmparray_r[BX], 1, &idx_max_bx);
    stat = hipblasIdamax(handle0, 2 * ntotal_complex, fields.d_tmparray_r[BY], 1, &idx_max_by);
    stat = hipblasIdamax(handle0, 2 * ntotal_complex, fields.d_tmparray_r[BZ], 1, &idx_max_bz);

    CUDA_RT_CALL(hipMemcpy(&maxbx, &fields.d_tmparray_r[BX][idx_max_bx-1], sizeof(scalar_type), hipMemcpyDeviceToHost));
    CUDA_RT_CALL(hipMemcpy(&maxby, &fields.d_tmparray_r[BY][idx_max_by-1], sizeof(scalar_type), hipMemcpyDeviceToHost));
    CUDA_RT_CALL(hipMemcpy(&maxbz, &fields.d_tmparray_r[BZ][idx_max_bz-1], sizeof(scalar_type), hipMemcpyDeviceToHost));
    // maxfx=d_farray_r[0][idx_max_vx-1];
    // maxfy=d_farray_r[1][idx_max_vy-1];
    // maxfz=d_farray_r[2][idx_max_vz-1];

    maxbx=fabs(maxbx);
    maxby=fabs(maxby);
    maxbz=fabs(maxbz);

    // std::printf("maxbx: %.5f \n",maxfx);
    // std::printf("maxby: %.5f \n",maxfy);
    // std::printf("maxbz: %.5f \n",maxfz);


    gamma_b = ( fields.wavevector.kxmax ) * maxbx + fields.wavevector.kymax * maxby + fields.wavevector.kzmax * maxbz;


#ifdef WITH_EXPLICIT_DISSIPATION
    gamma_b += ((fields.wavevector.kxmax )*( fields.wavevector.kxmax )+fields.wavevector.kymax*fields.wavevector.kymax+fields.wavevector.kzmax*fields.wavevector.kzmax) * param.nu_m;	// CFL condition on resistivity
#endif

#ifdef DDEBUG
    if (current_step == 1 || current_step % 100 == 0 ) std::printf("maxbx: %.4e \t maxby: %.4e \t maxbz: %.4e \t gamma_b: %.4e \n",maxbx,maxby,maxbz,gamma_b);
#endif

    dt = param.cfl / (gamma_v  + gamma_b);

    // r2c_fft(d_farray_r[BX], d_farray[BX]);
    // r2c_fft(d_farray_r[BY], d_farray[VY]);
    // r2c_fft(d_farray_r[BZ], d_farray[BZ]);

#else //not MHD

    dt = param.cfl / (gamma_v );

#endif //end MHD

    // r2c_fft(d_farray_r[VX], d_farray[VX]);
    // r2c_fft(d_farray_r[VY], d_farray[VY]);
    // r2c_fft(d_farray_r[VZ], d_farray[VZ]);

#endif //end INCOMPRESSIBLE

#ifdef HEAT_EQ
    double gamma_v = ((fields.wavevector.kxmax )*( fields.wavevector.kxmax )+fields.wavevector.kymax*fields.wavevector.kymax+fields.wavevector.kzmax*fields.wavevector.kzmax) * param.nu_th;

    dt = param.cfl / (gamma_v );
#endif

#ifdef DDEBUG
    if (current_step == 1 || current_step % 100 == 0 ) std::printf("t: %.4e \t dt: %.4e \n", current_time, dt);
#endif

    current_dt = dt;
    // *p_dt = dt;
}







    // absolute3<scalar_type>        unary_op;
    // thrust::maximum<scalar_type> binary_op;
    // maxfx = thrust::reduce(thrust::device_pointer_cast(d_farray_r[0]), thrust::device_pointer_cast(d_farray_r[0]) + 2*ntotal_complex, (double) 0, thrust::maximum<double>());
    // maxfx = thrust::transform_reduce(thrust::device_pointer_cast(d_farray_r[0]), thrust::device_pointer_cast(d_farray_r[0]) + 2*ntotal_complex, unary_op, (double) 0, binary_op);

    // old code that zips vx vy vz into a vector
    // Tuple3 temp;
    // auto begin = thrust::make_zip_iterator(thrust::make_tuple(thrust::device_pointer_cast(d_farray_r[0]),thrust::device_pointer_cast(d_farray_r[1]), thrust::device_pointer_cast(d_farray_r[2])));
    // auto end = thrust::make_zip_iterator(thrust::make_tuple(thrust::device_pointer_cast(d_farray_r[0]) + 2*ntotal_complex,thrust::device_pointer_cast(d_farray_r[1]) + 2*ntotal_complex, thrust::device_pointer_cast(d_farray_r[2]) + 2*ntotal_complex));
    // then finds 3-tuple with max vals
    // temp = thrust::transform_reduce(begin, end, absolute3<scalar_type>(), thrust::make_tuple<scalar_type,scalar_type,scalar_type>(0,0,0), MaxAbs<scalar_type>());
    // retrieve values
    // maxfx=thrust::get<0>(temp);
    // maxfy=thrust::get<1>(temp);
    // maxfz=thrust::get<2>(temp);


    // for timing
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // float milliseconds;
    //
    // int Niter = 100;
    // hipEventRecord(start);
    // for (int ii = 0; ii < Niter; ii++) {
    //     temp = thrust::transform_reduce(begin, end, absolute3<scalar_type>(), thrust::make_tuple<scalar_type,scalar_type,scalar_type>(0,0,0), MaxAbs<scalar_type>());
    // }
    // hipEventRecord(stop);
    // hipDeviceSynchronize();
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::printf("THRUST elapsed time (in s): %.5f \t Approx time per reduce (in ms): %.5f \n",milliseconds/1000, milliseconds/Niter);


    // equivalent code with CUBLAS (similar speed)
    // std::printf("now with CUBLAS \n");
    // int idx_max_vx, idx_max_vy, idx_max_vz;
    // hipblasStatus_t stat;
    //
    // // int Niter = 100;
    // hipEventRecord(start);
    // for (int ii = 0; ii < Niter; ii++) {
    //     stat = hipblasIdamax(handle0, 2 * ntotal_complex, d_farray_r[0], 1, &idx_max_vx);
    //     stat = hipblasIdamax(handle0, 2 * ntotal_complex, d_farray_r[1], 1, &idx_max_vy);
    //     stat = hipblasIdamax(handle0, 2 * ntotal_complex, d_farray_r[2], 1, &idx_max_vz);
    // }
    // hipEventRecord(stop);
    // hipDeviceSynchronize();
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::printf("CUBLAS elapsed time (in s): %.5f \t Approx time per reduce (in ms): %.5f \n",milliseconds/1000, milliseconds/Niter);
    // std::printf("idx_max_vx: %d \n",idx_max_vx);
    // std::printf("idx_max_vy: %d \n",idx_max_vy);
    // std::printf("idx_max_vz: %d \n",idx_max_vz);
    //
    // maxfx=d_farray_r[0][idx_max_vx];
    // maxfy=d_farray_r[1][idx_max_vy];
    // maxfz=d_farray_r[2][idx_max_vz];
