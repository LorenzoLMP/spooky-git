#include "define_types.hpp"
#include "cufft_routines.hpp"
#include "spooky.hpp"
#include "common.hpp"
#include "cuda_kernels.hpp"
#include "cuda_kernels_generic.hpp"
#include "timestepping.hpp"
#include "parameters.hpp"
#include "fields.hpp"
#include "physics.hpp"

void TimeStepping::compute_dfield(Fields &fields, Parameters &param, Physics &phys) {
    NVTX3_FUNC_RANGE();

    int blocksPerGrid;
    /*
     * Do all computations
     * required to compute dfield
     *
     */
#ifdef DDEBUG
    std::printf("Now entering compute_dfield function \n");
#endif


#ifdef HEAT_EQ

    // dT = nu_th nabla T
    //   #ifndef ANISOTROPIC_DIFFUSION
        // dfldo.th[i] += - nu_th * k2t[i] * fldi.th[i];
    //   #endif
    // computes nabla operator of T and assigns to dT
    // int blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    // nablaOp<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *) wavevector.d_kvec[KX],  (scalar_type *) wavevector.d_kvec[KY], (scalar_type *) wavevector.d_kvec[KZ], (hipfftDoubleComplex *) d_farray[TH], (hipfftDoubleComplex *) d_dfarray[TH], param.nu_th, (size_t) ntotal_complex, ASS);

    if (stage_step == 0) compute_dt(fields, param, phys);

    // laplacianScalar((scalar_type **)wavevector.d_kvec, (hipfftDoubleComplex *) d_farray[TH], (hipfftDoubleComplex *) d_dfarray[TH], param.nu_th, ASS);
    blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    // nablaOp<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *) wavevector.d_kvec[KX],  (scalar_type *) wavevector.d_kvec[KY], (scalar_type *) wavevector.d_kvec[KZ], (hipfftDoubleComplex *) d_farray[TH], (hipfftDoubleComplex *) d_dfarray[TH], param.nu_th, (size_t) ntotal_complex, ASS);
    nablaOpScalar<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *) fields.wavevector.d_all_kvec, (data_type *) fields.d_farray[TH], (data_type *) fields.d_dfarray[TH], param.nu_th, (size_t) ntotal_complex, ASS);

#endif

    // advectFields((scalar_type **)wavevector.d_kvec, (hipfftDoubleComplex **) d_farray, (hipfftDoubleComplex **) d_dfarray, ASS);

    // we are assuming that the first 3 arrays are always vx vy vz
    // for (int n = 0 ; n < num_fields ; n++) {
    //     c2r_fft(array_input[n], scratch[n]);
    // }
#ifdef INCOMPRESSIBLE

     // assign fields to [num_fields] tmparray (memory block starts at d_all_tmparray)
    blocksPerGrid = ( fields.num_fields * ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    ComplexVecAssign<<<blocksPerGrid, threadsPerBlock>>>((hipfftDoubleComplex *)fields.d_all_fields, (hipfftDoubleComplex *)fields.d_all_tmparray, fields.num_fields * ntotal_complex);

    // compute FFTs from complex to real fields to start computation of shear traceless matrix
    for (int n = 0; n < fields.num_fields; n++){
        c2r_fft(fields.d_tmparray[n], fields.d_tmparray_r[n], supervisor);
    }

    // hipDeviceSynchronize();
    if (stage_step == 0) compute_dt(fields, param, phys);

    // we use Basdevant formulation [1983]
    // compute the elements of the traceless symmetric matrix B_ij = u_i u_j - delta_ij Tr (u_i u_j) / 3. It has only 5 independent components B_xx, B_xy, B_xz, Byy, B_yz. (B_zz = - B_xx - B_yy)
    // the results are saved in the temp_arrays from [num_fields -- num_fields + 5] (the first num_fields arrays are reserved for the real-valued fields)
    blocksPerGrid = ( 2 * ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
#ifndef MHD
    TracelessShearMatrix<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)fields.d_all_tmparray, (scalar_type *)fields.d_all_tmparray + 2 * ntotal_complex * fields.num_fields,  2 * ntotal_complex);
#else
    TracelessShearMatrixMHD<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)fields.d_all_tmparray, (scalar_type *)fields.d_all_tmparray + 2 * ntotal_complex * fields.num_fields,  2 * ntotal_complex);
#endif


    // take fft of 5 independent components of B_ij
    for (int n = fields.num_fields ; n < fields.num_fields + 5; n++) {
        r2c_fft(fields.d_tmparray_r[n], fields.d_tmparray[n], supervisor);
    }

    // compute derivative of traceless shear matrix and assign to dfields
    // this kernel works also if MHD
    blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    NonLinHydroAdv<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)fields.wavevector.d_all_kvec, (data_type *)fields.d_all_tmparray + ntotal_complex * fields.num_fields, (data_type *) fields.d_all_dfields, (scalar_type *)fields.wavevector.d_mask, ntotal_complex);


#ifdef MHD
    // compute emf = u x B:
    // emf_x = u_y B_z - u_z B_y , emf_y = u_z B_x - u_x B_z , emf_z = u_x B_y - u_y B_x
    // the results are saved in the first 3 temp_arrays (after those reserved for the fields, the memory block points already at the right location) as [emf_x, emf_y, emf_z] (they are the x,y,z components of the emf)
    // We can re-utilize tmparrays and store result in tmparray_r[num_fields] - tmparray_r[num_fields + 3]
    blocksPerGrid = ( 2 * ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    MagneticEmf<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)fields.d_all_tmparray, (scalar_type *)fields.d_all_tmparray + 2 * ntotal_complex * fields.num_fields,  2 * ntotal_complex);

    // take fourier transforms of the 3 independent components of the antisymmetric shear matrix
    for (int n = fields.num_fields ; n < fields.num_fields + 3; n++) {
        r2c_fft(fields.d_tmparray_r[n], fields.d_tmparray[n], supervisor);
    }

    // compute derivative of antisymmetric magnetic shear matrix and assign to dfields
    blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    MagneticShear<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)fields.wavevector.d_all_kvec, (data_type *)fields.d_all_tmparray + ntotal_complex * fields.num_fields, (data_type *) fields.d_all_dfields, (scalar_type *)fields.wavevector.d_mask, ntotal_complex);



#endif

#ifdef BOUSSINESQ
    // This function assumes that the real transforms of the fields are stored in tmparrays_r[0] - tmparray_r[num_fields - 1]
    phys.Boussinesq(fields, param);
#endif

// #ifdef BOUSSINESQ
//     // for hydro-Boussinesq computation of u nabla theta can go here
//
//     // first compute energy flux vector [ u_x theta, u_y theta, u_z theta]
//     // we can re-utilize tmparrays and store result in tmparray_r[num_fields] - tmparray_r[num_fields + 3]
//     blocksPerGrid = ( 2 * ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
//     EnergyFluxVector<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)d_all_tmparray, (scalar_type *)d_all_tmparray + 2 * ntotal_complex * num_fields,  2 * ntotal_complex);
//
//     // scalar_type *host_tmp;
//     // host_tmp = (scalar_type *) malloc( (size_t) sizeof(scalar_type) * 2 * ntotal_complex );
//     // for (int i = 0; i < 2 * ntotal_complex ; i++){
//     //     host_tmp[i] = 0.0;
//     // }
//     // CUDA_RT_CALL(hipMemcpy(host_tmp, (scalar_type*)d_tmparray_r[4], sizeof(scalar_type) * 2 * ntotal_complex , hipMemcpyDeviceToHost));
//     // unsigned int idx;
//     // for (int i = 25; i < 32; i++){
//     //     idx =  (nz/2+1)*2 * ( i * ny);
//     //     // std::printf("v1[%d]= %f \t v2[%d]= %f \n", idx, farray_r[0][idx], idx, farray_r[1][idx]);
//     //     for (int n = 0; n < 1; n++){
//     //         std::printf("tmp[%d][%d] = %.3e \t", n, idx, host_tmp[idx]);
//     //     }
//     //     std::cout << std::endl;
//     // }
//     // for (int i = 0; i < 2 * ntotal_complex ; i++){
//     //     if (host_tmp[i] != 0.0) {
//     //         std::printf("BREAK:   tmp[%d] = %.3e \n", i, host_tmp[i]);
//     //         break;
//     //     }
//     // }
//     // free(host_tmp);
//
//     // take fourier transforms of the 3 energy flux vector components
//     for (int n = num_fields ; n < num_fields + 3; n++) {
//         r2c_fft(d_tmparray_r[n], d_tmparray[n]);
//     }
//
//     // scalar_type *host_tmp;
//     // host_tmp = (scalar_type *) malloc( (size_t) sizeof(scalar_type) * 2 * ntotal_complex );
//     // for (int i = 0; i < 2 * ntotal_complex ; i++){
//     //     host_tmp[i] = 0.0;
//     // }
//     // CUDA_RT_CALL(hipMemcpy(host_tmp, (scalar_type*)d_tmparray[4], sizeof(scalar_type) * 2 * ntotal_complex , hipMemcpyDeviceToHost));
//     // unsigned int idx;
//     // for (int i = 0; i < 25; i++){
//     //     idx =  (nz/2+1)*2 * ( i * ny);
//     //     // std::printf("v1[%d]= %f \t v2[%d]= %f \n", idx, farray_r[0][idx], idx, farray_r[1][idx]);
//     //     for (int n = 0; n < 1; n++){
//     //         std::printf("tmp[%d][%d] = %.3e \t", n, idx, host_tmp[idx]);
//     //     }
//     //     std::cout << std::endl;
//     // }
//     // free(host_tmp);
//
//     // compute derivative of energy flux vector and assign u nabla theta to the dfield for theta
//     blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
//     NonLinBoussinesqAdv<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)wavevector.d_all_kvec, (data_type *)d_all_tmparray + ntotal_complex * num_fields, (data_type *) d_all_dfields, (scalar_type *)wavevector.d_mask, ntotal_complex);
//
//
//
// #ifdef STRATIFICATION
//     EntropyStratification();
//     // add - th e_strat to velocity component in the strat direction
//     // add N2 u_strat to temperature equation
//     // this is for normalization where theta is in units of g [L/T^2]
//     // other normalizations possible
//     // blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
//     // BoussinesqStrat<<<blocksPerGrid, threadsPerBlock>>>( (data_type *)d_all_fields, (data_type *) d_all_dfields, param.N2, ntotal_complex, STRAT_DIR);
// #endif
//
// #ifndef ANISOTROPIC_DIFFUSION
//     //  for explicit treatment of energy diffusion term
//     blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
//     nablaOpScalar<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *) wavevector.d_all_kvec, (data_type *) d_farray[TH], (data_type *) d_dfarray[TH], param.nu_th, (size_t) ntotal_complex, ADD);
// #else
// #ifdef MHD
//     AnisotropicConduction();
//     /*
//     // assign Bx, By, Bz to first 3 scratch arrays
//     blocksPerGrid = ( 3 * ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
//     ComplexVecAssign<<<blocksPerGrid, threadsPerBlock>>>((hipfftDoubleComplex *)d_all_fields + ntotal_complex * BX, (hipfftDoubleComplex *)d_all_tmparray, 3 * ntotal_complex);
//     // compute gradient of theta and assign it to next 3 scratch arrays
//     blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
//     Gradient<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)wavevector.d_all_kvec, (data_type *) d_farray[TH], (data_type *)d_all_tmparray + 3 * ntotal_complex, ntotal_complex);
//     // compute complex to real iFFTs
//     for (int n = 0; n < 6; n++){
//         c2r_fft(d_tmparray[n], d_tmparray_r[n]);
//     }
//     // compute the scalar B grad theta (real space) and assign it to 7th scratch array
//     blocksPerGrid = ( 2 * ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
//     ComputeBGradTheta<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *) d_tmparray_r[0], (scalar_type *) d_tmparray_r[3], (scalar_type *) d_tmparray_r[6], 2 * ntotal_complex);
//     // compute the anisotropic heat flux and put it in the 3-4-5 tmp arrays
//     ComputeAnisotropicHeatFlux<<<blocksPerGrid, threadsPerBlock>>>( (scalar_type *) d_tmparray_r[0], (scalar_type *) d_tmparray_r[6], (scalar_type *) d_tmparray_r[3], param.OmegaT2, (1./param.reynolds_ani), 2 * ntotal_complex, STRAT_DIR);
//     // take fourier transforms of the heat flux
//     for (int n = 3 ; n < 6; n++) {
//         r2c_fft(d_tmparray_r[n], d_tmparray[n]);
//     }
//     // take divergence of heat flux
//     blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
//     DivergenceMask<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)wavevector.d_all_kvec, (data_type *) d_tmparray[3], (data_type *) d_all_dfields + TH * ntotal_complex, (scalar_type *)wavevector.d_mask, ntotal_complex, ADD);
//     */
// #endif   // MHD
// #endif   // ANISOTROPIC_DIFFUSION
//
//
// #endif // Boussinesq

/*
 *
 * Now we enforce the incompressibility
 * condition
 *
 */


    // compute pseudo-pressure and subtract grad p_tilde from dfields
    blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    GradPseudoPressure<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *)fields.wavevector.d_all_kvec, (data_type *) fields.d_all_dfields, ntotal_complex);


/*
 *
 * Here we do the diffusion terms
 *
 */

    // for explicit treatment of diffusion terms
    // with incompressible d_all_fields always points at VX
    blocksPerGrid = ( ntotal_complex + threadsPerBlock - 1) / threadsPerBlock;
    nablaOpVector<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *) fields.wavevector.d_all_kvec, (data_type *) fields.d_all_fields, (data_type *) fields.d_all_dfields, param.nu, (size_t) ntotal_complex, ADD);

#ifdef MHD
    // for explicit treatment of diffusion terms
    // point d_all_fields at BX
    nablaOpVector<<<blocksPerGrid, threadsPerBlock>>>((scalar_type *) fields.wavevector.d_all_kvec, (data_type *) fields.d_all_fields + ntotal_complex * BX, (data_type *) fields.d_all_dfields + ntotal_complex * BX, param.nu_m, (size_t) ntotal_complex, ADD);
#endif



    // will use temp arrays to store data d_tmparray_r and d_tmparray
    // for (int n = 0 ; n < num_fields ; n++) {
    //     c2r_fft(d_farray[n], d_tmparray_r[n]);
    // }
    //
    //
    //
    // for (int n = 0 ; n < num_fields ; n++) {
    //     r2c_fft(d_farray_r[n], d_farray[n]);
    // }


#endif //end INCOMPRESSIBLE



}


