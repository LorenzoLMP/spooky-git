#include "define_types.hpp"
#include "timestepping.hpp"
// #include "cufft_routines.hpp"
#include "spooky.hpp"
#include "common.hpp"
#include "cublas_routines.hpp"
#include "cuda_kernels.hpp"
#include "parameters.hpp"
// #include "inputoutput.hpp"
#include "fields.hpp"
#include <hip/hip_runtime.h>
// #include <hipfft/hipfftXt.h>
// #include "spooky.hpp"
#include "cufft_utils.h"
// #include "define_types.hpp"
#include "supervisor.hpp"

TimeStepping::TimeStepping(int num, Supervisor &sup) {
    // param = &p_in;
    // fields = &f_in;

    supervisor = &sup;
    // std::printf("The TimeSpentInFFTs is: %.4e",supervisor->TimeSpentInFFTs);
    current_dt = 0.0;
    current_time = 0.0;
    current_step = 0;
    // stage_step = 0;

    // this is the mega array that contains intermediate fields during multi-stage timestepping
    // std::printf("num fields ts: %d \n", fields->num_fields);
    std::printf("num timestepping scratch arrays: %d \n",num);
    CUDA_RT_CALL(hipMalloc(&d_all_scrtimestep, (size_t) sizeof(data_type) * ntotal_complex * num));
}

TimeStepping::~TimeStepping(){
    CUDA_RT_CALL(hipFree(d_all_scrtimestep));
}
