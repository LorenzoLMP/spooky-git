#include "spooky.hpp"
#include <hip/hip_runtime.h>
// #include <hipfft/hipfftXt.h>
#include "cufft_utils.h"
#include "cufft_routines.hpp"
#include "cublas_routines.hpp"
#include "tests.hpp"
#include <complex.h>
#include "fields.hpp"
// #include "wavevector.hpp"
#include "common.hpp"
// #include "parameters.hpp"
// #include "cuda_kernels.hpp"
// #include "cuda_kernels_generic.hpp"
#include <argparse/argparse.hpp>
#include "parameters.hpp"
#include "inputoutput.hpp"
#include "timestepping.hpp"
#include "physics.hpp"
#include "supervisor.hpp"

void startup();

Variables vars;
Grid grid;

int main(int argc, char *argv[]) {

    int restart_num = -1;
    // int stats_frequency = -1;

    argparse::ArgumentParser program("spooky");

    program.add_argument("--input-dir")
    .help("input directory for cfg file")
    .default_value(std::string("./"));

    program.add_argument("--output-dir")
    .help("output directory for data files");

    program.add_argument("-r", "--restart")
    .help("restart from data file")
    .scan<'i', int>();
    // .default_value(int(-1));

    program.add_argument("--stats")
    .help("whether to print stats: -1 (none), n > 0 (every n steps)")
    .scan<'i', int>()
    .default_value(int(-1));


    try {
    program.parse_args(argc, argv);
    }
    catch (const std::exception& err) {
    std::cerr << err.what() << std::endl;
    std::cerr << program;
    std::exit(1);
    }

    std::string input_dir = program.get<std::string>("--input-dir");
    std::cout << "Input directory: " << input_dir << std::endl;



    startup();
    
    std::printf("-----------Initializing cufft, cublas...\n");

    init_plan(fft_size);
    init_cublas();

    //----------------------------------------------------------------------------------------
    //! Initialize objects

    std::printf("-----------Initializing objects...\n");

    Supervisor spooky(input_dir);
    // Supervisor spooky(stats_frequency);
    //
    // Parameters param(input_dir);
    // Fields fields(param, NUM_FIELDS);
    // Physics phys(spooky);
    // TimeStepping timestep(NUM_FIELDS, param, spooky);
    // InputOutput inout(spooky);


    std::printf("Finished reading in params and initializing objects.\n");


    //----------------------------------------------------------------------------------------
    //! Parse runtime flags and override default params

    if (program.is_used("--output-dir")){
        std::string output_dir = program.get<std::string>("--output-dir");
        std::cout << "output directory will be overriden: " << output_dir << std::endl;
        spooky.param_ptr->output_dir = output_dir;
    }
    if (program.is_used("--restart")){
        // std::cout << "restarting from file: "  << std::endl;
        restart_num = program.get<int>("--restart");
        std::cout << "restarting from file: " << restart_num << std::endl;
        spooky.param_ptr->restart = 1;
    }

    if (program.is_used("--stats")){
        spooky.stats_frequency = program.get<int>("--stats");
        std::cout << "printing stats every " << spooky.stats_frequency << " steps " << std::endl;
    }

    spooky.displayConfiguration();

    spooky.Restart(restart_num);

    if (spooky.param_ptr->debug > 1) {
        spooky.fields_ptr->wavevector.print_values();
        spooky.fields_ptr->print_host_values();
    }

    std::printf("Allocating to gpu...\n");
    spooky.fields_ptr->allocate_and_move_to_gpu();

    spooky.fields_ptr->CheckSymmetries();

    spooky.initialDataDump();

    spooky.executeMainLoop();

    spooky.print_final_stats();

    // std::printf("Starting copy back to host\n");
    spooky.fields_ptr->copy_back_to_host();
    

    spooky.fields_ptr->clean_gpu();
    std::printf("Finished fields gpu cleanup\n");

    if (spooky.param_ptr->debug > 1) {
        spooky.fields_ptr->print_host_values();
    }

    std::printf("Finishing cufft\n");
    finish_cufft();

    std::printf("Finishing cublas\n");
    finish_cublas();

    return EXIT_SUCCESS;
};


void startup(){
	std::cout <<
R"abcd(
                 ____________
               --            --
             /                  \\
            /                    \\
           /     __               \\
          |     /  \       __      ||
          |    |    |     /  \     ||
                \__/      \__/
         |             ^            ||
         |                          ||
         |                          ||
        |                            ||
        |                            ||
        |                            ||
         \__         ______       __//
            \       //     \_____//
             \_____//

)abcd" << std::endl;
}



