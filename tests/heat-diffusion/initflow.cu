#include "hip/hip_runtime.h"
#include "common.hpp"
#include "cufft_routines.hpp"
// #include "cuda_kernels.hpp"
#include "fields.hpp"
#include "parameters.hpp"
#include "supervisor.hpp"

void Fields::initSpatialStructure(){

	int i,j,k;
	std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;

	/*******************************************************************
	** This part does not need to be modified **************************
	********************************************************************/
	// Allocate coordinate arrays
	scalar_type *x, *y, *z;
	// cpudata_t x((size_t) grid.NTOTAL_COMPLEX);
    // cpudata_t y((size_t) grid.NTOTAL_COMPLEX);
    // cpudata_t z((size_t) grid.NTOTAL_COMPLEX);

	x = (scalar_type *) malloc( (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX);
	y = (scalar_type *) malloc( (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX);
	z = (scalar_type *) malloc( (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX);
 //
 //    // Initialize the arrays
	// // MPI_Printf("grid.NZ = %d \n", grid.NZ);
    #ifndef WITH_2D
	for(i = 0 ; i < grid.NX ; i++) {
		for(j = 0 ; j < grid.NY ; j++) {
			for(k = 0 ; k < grid.NZ ; k++) {
				x[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = - param_ptr->lx / 2 + (param_ptr->lx * i) / grid.NX;
				y[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = - param_ptr->ly / 2 + (param_ptr->ly * j ) / grid.NY;
				z[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = - param_ptr->lz / 2 + (param_ptr->lz * k ) / grid.NZ;
			}
		}
		// std::printf("x[%d] = %.2e \t",i,x[(grid.NZ + 2) * grid.NY * i]);
	}
	std::cout << std::endl;
	// std::printf("coords initialized\n");
	#else
	for(i = 0 ; i < grid.NX ; i++) {
		for(j = 0 ; j < grid.NY ; j++) {
			for(k = 0 ; k < grid.NZ ; k++) {
				x[k + (grid.NZ) * j + (grid.NZ) * (grid.NY + 2) * i] = - param_ptr->lx / 2 + (param_ptr->lx * i) / grid.NX;
				y[k + (grid.NZ) * j + (grid.NZ) * (grid.NY + 2) * i] = - param_ptr->ly / 2 + (param_ptr->ly * j ) / grid.NY;
				z[k + (grid.NZ) * j + (grid.NZ) * (grid.NY + 2) * i] = - param_ptr->lz / 2 + (param_ptr->lz * k ) / grid.NZ;
			}
		}
	}
    #endif
	// Initialize the extra points (k=grid.NZ and k=grid.NZ+1) to zero to prevent stupid things from happening...
	#ifndef WITH_2D
	for(i = 0 ; i < grid.NX ; i++) {
		for(j = 0 ; j < grid.NY ; j++) {
			for(k = grid.NZ ; k < grid.NZ + 2 ; k++) {
				x[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = 0.0;
				y[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = 0.0;
				z[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = 0.0;
			}
		}
	}
	#else
	for(i = 0 ; i < grid.NX ; i++) {
		for(j = grid.NY ; j < grid.NY + 2; j++) {
			for(k = 0 ; k < grid.NZ ; k++) {
				x[k + (grid.NZ ) * j + (grid.NZ ) * (grid.NY + 2) * i] = 0.0;
				y[k + (grid.NZ ) * j + (grid.NZ ) * (grid.NY + 2) * i] = 0.0;
				z[k + (grid.NZ ) * j + (grid.NZ ) * (grid.NY + 2) * i] = 0.0;
			}
		}
	}
	#endif


	///////////////////////////////////////
	// initial conditions on host data
	///////////////////////////////////////
	double a = 0.01;
	for (int i = 0; i < 2*grid.NTOTAL_COMPLEX; i++){

		// 1D heat diffusion
	// #ifdef HEAT_EQ
	farray_r[vars.TH][i] = 1.0 +  0.5 * (tanh((x[i] + 0.375) / a) - tanh((x[i] + 0.125) / a)) + 0.5 * (tanh((x[i] - 0.125) / a) - tanh((x[i] - 0.375) / a));
	// #endif

	}

// 	for (int i = 0; i < 10; i++){
//
// 		std::printf("x[%d] = %.2e \t y[%d] = %.2e \t z[%d] = %.2e \t th[%d] = %.2e \t",i,x[i],i,y[i],i,z[i],i,farray_r[vars.TH][i]);
//
// 	}

	// int idx;
	// #ifdef HEAT_EQ
	// for(i = 0 ; i < grid.NX ; i++) {
	// 	idx = (grid.NZ + 2) * grid.NY * i;
	// 	std::printf("x[%d] = %.2e \t  th[%d] = %.2e \n",idx,x[idx],idx,farray_r[vars.TH][idx]);
	// }
	// #endif
 //
 //
 //

	free(x);
	free(y);
	free(z);

	std::printf("Finished initializing spatial structure\n");
}
