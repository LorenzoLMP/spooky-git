#include "hip/hip_runtime.h"
// #include "define_types.hpp"
// // #include "fields.hpp"
// #include "cufft_routines.hpp"
#include "spooky.hpp"
#include "common.hpp"
// #include "fields.hpp"
// #include "parameters.hpp"
#include "user_outputs.hpp"
#include "cublas_routines.hpp"
#include "cuda_kernels.hpp"
#include "cuda_kernels_generic.hpp"
#include "cufft_routines.hpp"
#include "user_defined_cuda_kernels.hpp"

UserOutput::UserOutput() {
    // double lx, ly, lz;
    // read_Parameters();
}

UserOutput::~UserOutput() {
}


scalar_type UserOutput::customFunction( data_type *vcomplex ) {
    /***
     * This function uses complex input to compute the "energy"
     * The modes with k>0 only have half the energy (because the k<0 is not present).
     * Here we multiply all k modes by 2 and then subtract once the energy in the k=0 mode.
     * The total is then divided by 2 to give quantity (i.e. Energy ~ (1/2) v^2)
     ***/

    // hipblasStatus_t stat;
    // scalar_type norm = 0.0;
    scalar_type var = 0.0;
    // scalar_type subtract = 0.0;
    // scalar_type tmp = 0.0;

    return var;
}
