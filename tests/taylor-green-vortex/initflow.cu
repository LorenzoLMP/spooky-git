#include "hip/hip_runtime.h"
#include "common.hpp"
#include "cufft_routines.hpp"
// #include "cuda_kernels.hpp"
#include "fields.hpp"
#include "parameters.hpp"
#include "supervisor.hpp"

void Fields::initSpatialStructure(){

	std::shared_ptr<Parameters> param_ptr = supervisor_ptr->param_ptr;

	int i,j,k;

	/*******************************************************************
	** This part does not need to be modified **************************
	********************************************************************/
	// Allocate coordinate arrays
	scalar_type *x, *y, *z;
	// cpudata_t x((size_t) grid.NTOTAL_COMPLEX);
    // cpudata_t y((size_t) grid.NTOTAL_COMPLEX);
    // cpudata_t z((size_t) grid.NTOTAL_COMPLEX);

	x = (scalar_type *) malloc( (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX);
	y = (scalar_type *) malloc( (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX);
	z = (scalar_type *) malloc( (size_t) sizeof(data_type) * grid.NTOTAL_COMPLEX);
 //
 //    // Initialize the arrays
	// // MPI_Printf("grid.NZ = %d \n", grid.NZ);
    #ifndef WITH_2D
	for(i = 0 ; i < grid.NX ; i++) {
		for(j = 0 ; j < grid.NY ; j++) {
			for(k = 0 ; k < grid.NZ ; k++) {
				x[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = - param_ptr->lx / 2 + (param_ptr->lx * i) / grid.NX;
				y[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = - param_ptr->ly / 2 + (param_ptr->ly * j ) / grid.NY;
				z[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = - param_ptr->lz / 2 + (param_ptr->lz * k ) / grid.NZ;
			}
		}
		// std::printf("x[%d] = %.2e \t",i,x[(grid.NZ + 2) * grid.NY * i]);
	}
	std::cout << std::endl;
	// std::printf("coords initialized\n");
	#else
	for(i = 0 ; i < grid.NX ; i++) {
		for(j = 0 ; j < grid.NY ; j++) {
			for(k = 0 ; k < grid.NZ ; k++) {
				x[k + (grid.NZ) * j + (grid.NZ) * (grid.NY + 2) * i] = - param_ptr->lx / 2 + (param_ptr->lx * i) / grid.NX;
				y[k + (grid.NZ) * j + (grid.NZ) * (grid.NY + 2) * i] = - param_ptr->ly / 2 + (param_ptr->ly * j ) / grid.NY;
				z[k + (grid.NZ) * j + (grid.NZ) * (grid.NY + 2) * i] = - param_ptr->lz / 2 + (param_ptr->lz * k ) / grid.NZ;
			}
		}
	}
    #endif
	// Initialize the extra points (k=grid.NZ and k=grid.NZ+1) to zero to prevent stupid things from happening...
	#ifndef WITH_2D
	for(i = 0 ; i < grid.NX ; i++) {
		for(j = 0 ; j < grid.NY ; j++) {
			for(k = grid.NZ ; k < grid.NZ + 2 ; k++) {
				x[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = 0.0;
				y[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = 0.0;
				z[k + (grid.NZ + 2) * j + (grid.NZ + 2) * grid.NY * i] = 0.0;
			}
		}
	}
	#else
	for(i = 0 ; i < grid.NX ; i++) {
		for(j = grid.NY ; j < grid.NY + 2; j++) {
			for(k = 0 ; k < grid.NZ ; k++) {
				x[k + (grid.NZ ) * j + (grid.NZ ) * (grid.NY + 2) * i] = 0.0;
				y[k + (grid.NZ ) * j + (grid.NZ ) * (grid.NY + 2) * i] = 0.0;
				z[k + (grid.NZ ) * j + (grid.NZ ) * (grid.NY + 2) * i] = 0.0;
			}
		}
	}
	#endif


	///////////////////////////////////////
	// initial conditions on host data
	///////////////////////////////////////
	for (int i = 0; i < 2*grid.NTOTAL_COMPLEX; i++){

		// Taylor - Green vortex
		// farray_r[vars.VX][i] =   sin(2.0*M_PI*x[i]/param_ptr->lx) * cos(2.0*M_PI*y[i]/param_ptr->ly);
		// // farray_r[vars.VY][i] = 1.0 * sin(2.0*M_PI*x[i]);
		// farray_r[vars.VY][i] = - cos(2.0*M_PI*x[i]/param_ptr->lx) * sin(2.0*M_PI*y[i]/param_ptr->ly);
		// farray_r[vars.VZ][i] = 0.0;

		farray_r[vars.VX][i] = 0.0 ;
		// farray_r[vars.VY][i] = 1.0 * sin(2.0*M_PI*x[i]);
		farray_r[vars.VY][i] = sin(2.0*M_PI*y[i]/param_ptr->ly) * cos(2.0*M_PI*z[i]/param_ptr->lz) ;
		farray_r[vars.VZ][i] = - cos(2.0*M_PI*y[i]/param_ptr->ly) * sin(2.0*M_PI*z[i]/param_ptr->lz);

	}

// 	for (int i = 0; i < 10; i++){
//
// 		std::printf("x[%d] = %.2e \t y[%d] = %.2e \t z[%d] = %.2e \t th[%d] = %.2e \t",i,x[i],i,y[i],i,z[i],i,farray_r[vars.TH][i]);
//
// 	}

	// int idx;
	// #ifdef HEAT_EQ
	// for(i = 0 ; i < grid.NX ; i++) {
	// 	idx = (grid.NZ + 2) * grid.NY * i;
	// 	std::printf("x[%d] = %.2e \t  th[%d] = %.2e \n",idx,x[idx],idx,farray_r[vars.TH][idx]);
	// }
	// #endif
 //
 //
 //

	free(x);
	free(y);
	free(z);

	std::printf("Finished initializing spatial structure\n");
}
